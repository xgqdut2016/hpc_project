#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

double
get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}
void matrixSerial(float *hostA, float *hostB, float *hostC, int M, int K, int N)
{
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            float tmp = 0;
            for (int s = 0; s < K; s++)
            {
                tmp += hostA[i * K + s] * hostB[s * N + j];
            }
            hostC[i * N + j] = tmp;
        }
    }
}
void compare(float *hostC, float *serialC, int M, int N)
{
    float error = 0;
    bool tmp = true;
    for (int i = 0; i < M * N; i++)
    {
        error = fmax(error, fabs(hostC[i] - serialC[i]));
        if (error > 1e-5)
        {
            tmp = false;
            printf("error:hostC[%d] = %.3f, serialC[%d] = %.3f\n", i, hostC[i], i, serialC[i]);
            break;
        }
    }
    if (tmp)
    {
        printf("cublas output all right\n");
    }
}

void cublasMatrix(float *hostA, float *hostB, float *hostC, int M, int K, int N)
{
    double st, ela;
    st = get_walltime();

    float *dA, *dB, *dC;
    hipMalloc((void **)&dA, M * K * sizeof(float));
    hipMalloc((void **)&dB, N * K * sizeof(float));
    hipMalloc((void **)&dC, M * N * sizeof(float));

    hipMemcpy(dA, hostA, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hostB, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipblasHandle_t handle; // cublas句柄
    hipblasCreate(&handle); // 初始化句柄
    float alpha = 1.0;
    float beta = 0.0;
    int repeat = 20;
    // cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, M, K, &alpha, dB, N, dA, K, &beta, dC, N);
    // cublasSgemm(handle, CUBLAS_OP_T, CUBLAS_OP_T, M, N, K, &alpha, dA, K, dB, N, &beta, dC, M);
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 N, M, K,
                 &alpha,
                 dB, HIP_R_32F, N,
                 dA, HIP_R_32F, K,
                 &beta,
                 dC, HIP_R_32F, N,
                 HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < repeat; i++)
    {
        // cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, M, K, &alpha, dB, N, dA, K, &beta, dC, N);
        // cublasSgemm(handle, CUBLAS_OP_T, CUBLAS_OP_T, M, N, K, &alpha, dA, K, dB, N, &beta, dC, M);
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                     N, M, K,
                     &alpha,
                     dB, HIP_R_32F, N,
                     dA, HIP_R_32F, K,
                     &beta,
                     dC, HIP_R_32F, N,
                     HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop); // must float ker_time

    hipMemcpy(hostC, dC, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    ela = get_walltime() - st;
    printf("M-K-N: %d-%d-%d\n", M, K, N);
    printf("cublas time: %.4f second\n", ela);
    printf("kernel time: %.4f second, %.4f ms\n", ker_time / (repeat * 1000.), ker_time / repeat);
}

int main()
{
    float *hostA, *hostB, *hostC, *serialC;
    int M = 1024;
    int K = 1024;
    int N = 1024;

    hostA = (float *)malloc(M * K * sizeof(float));
    hostB = (float *)malloc(N * K * sizeof(float));
    hostC = (float *)malloc(M * N * sizeof(float));
    serialC = (float *)malloc(M * N * sizeof(float));
    for (int i = 0; i < M * K; i++)
    {
        hostA[i] = i % 3;
    }
    for (int i = 0; i < N * K; i++)
    {
        hostB[i] = i % 3;
    }
    cublasMatrix(hostA, hostB, hostC, M, K, N);
    double st, ela;
    st = get_walltime();
    matrixSerial(hostA, hostB, serialC, M, K, N);
    ela = get_walltime() - st;
    printf("CPU time:%.2f second\n", ela);
    compare(hostC, serialC, M, N);
    free(hostA);
    free(hostB);
    free(hostC);
    free(serialC);
    return 0;
}
