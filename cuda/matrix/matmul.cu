#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

double
get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}
void matrixSerial(float *hostA, float *hostB, float *hostC, int M, int K, int N)
{
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            float tmp = 0;
            for (int s = 0; s < K; s++)
            {
                tmp += hostA[i * K + s] * hostB[s * N + j];
            }
            hostC[i * N + j] = tmp;
        }
    }
}
float compare(float *hostC, float *serialC, int M, int N)
{
    float error = 0;
    for (int i = 0; i < M * N; i++)
    {
        error = fmax(error, fabs(hostC[i] - serialC[i]));
    }
    return error;
}
__global__ void matrixKernel1st(float *dA, float *dB, float *dC, int M, int K, int N)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    float tmp = 0;
    if (row < M && col < N)
    {
        for (int s = 0; s < K; s++)
        {
            tmp += dA[row * K + s] * dB[s * N + col];
        }
        dC[row * N + col] = tmp;
    }
}
template <int BLOCK_DIM>
__global__ void matrixKernel2nd(float *dA, float *dB, float *dC, int M, int K, int N)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    float tmp = 0.0f;
    __shared__ float SA[BLOCK_DIM][BLOCK_DIM];
    __shared__ float SB[BLOCK_DIM][BLOCK_DIM];
    int width = (K + BLOCK_DIM - 1) / BLOCK_DIM;
    for (int ph = 0; ph < width; ph++)
    {
        if (row < M && threadIdx.y + ph * BLOCK_DIM < K)
        {
            SA[threadIdx.x][threadIdx.y] = dA[row * K + threadIdx.y + ph * BLOCK_DIM];
        }
        else
        {
            SA[threadIdx.x][threadIdx.y] = 0.0f;
        }
        if (threadIdx.x + ph * BLOCK_DIM < K && col < N)
        {
            SB[threadIdx.x][threadIdx.y] = dB[(threadIdx.x + ph * BLOCK_DIM) * N + col];
        }
        else
        {
            SB[threadIdx.x][threadIdx.y] = 0.0f;
        }
        __syncthreads();
        for (int s = 0; s < BLOCK_DIM; s++)
        {
            tmp += SA[threadIdx.x][s] * SB[s][threadIdx.y];
        }
        __syncthreads();
    }
    if (row < M && col < N)
    {
        dC[row * N + col] = tmp;
    }
}

void hostMatrix(float *hostA, float *hostB, float *hostC, int M, int K, int N)
{
    double st, ela;
    st = get_walltime();

    float *dA, *dB, *dC;
    hipMalloc((void **)&dA, M * K * sizeof(float));
    hipMalloc((void **)&dB, N * K * sizeof(float));
    hipMalloc((void **)&dC, M * N * sizeof(float));

    hipMemcpy(dA, hostA, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hostB, N * K * sizeof(float), hipMemcpyHostToDevice);

    int BLOCK_DIM_x = 32;
    int BLOCK_DIM_y = 32;
    int num_blocks_x = (M + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
    int num_blocks_y = (N + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
    dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
    dim3 grid_dim(num_blocks_x, num_blocks_y, 1);
    int repeat = 20;
    // matrixKernel1st<<<grid_dim, block_dim>>>(dA, dB, dC, M, K, N);
    matrixKernel2nd<32><<<grid_dim, block_dim>>>(dA, dB, dC, M, K, N);
    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < repeat; i++)
    {
        // matrixKernel1st<<<grid_dim, block_dim>>>(dA, dB, dC, M, K, N);
        matrixKernel2nd<32><<<grid_dim, block_dim>>>(dA, dB, dC, M, K, N);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop); // must float ker_time

    hipMemcpy(hostC, dC, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    ela = get_walltime() - st;
    printf("M-K-N: %d-%d-%d\n", M, K, N);
    printf("GPU use time: %.4f second\n", ela);
    printf("kernel time: %.4f second, %.4f ms\n", ker_time / (repeat * 1000.), ker_time / repeat);
    printf("grid dim: %d, %d, %d\n", grid_dim.x, grid_dim.y, grid_dim.z);
    printf("block dim: %d, %d, %d\n", block_dim.x, block_dim.y, block_dim.z);
}

int main()
{
    float *hostA, *hostB, *hostC, *serialC;
    int M = 1024;
    int K = 1024;
    int N = 1024;

    hostA = (float *)malloc(M * K * sizeof(float));
    hostB = (float *)malloc(N * K * sizeof(float));
    hostC = (float *)malloc(M * N * sizeof(float));
    serialC = (float *)malloc(M * N * sizeof(float));
    for (int i = 0; i < M * K; i++)
    {
        hostA[i] = i % 3;
    }
    for (int i = 0; i < N * K; i++)
    {
        hostB[i] = i % 3;
    }
    hostMatrix(hostA, hostB, hostC, M, K, N);
    double st, ela;
    st = get_walltime();
    matrixSerial(hostA, hostB, serialC, M, K, N);
    ela = get_walltime() - st;
    float error = compare(hostC, serialC, M, N);
    printf("CPU time:%.2f second\n", ela);
    printf("The error between CPU and GPU: %.4e\n", error);
    free(hostA);
    free(hostB);
    free(hostC);
    free(serialC);
    return 0;
}
