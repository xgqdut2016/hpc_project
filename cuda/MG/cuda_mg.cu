#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>

#define BLOCK_DIM ((int)8)
extern "C" void cuda_jacobi_solve(int epoch, double eps,double bound[][2], double *u_old, double *u_new, double *b, double *resid, int M, int N);
extern "C" double cuda_norm(double *cuda_u, int M, int N);
extern "C" void device_jacobi(int epoch, double eps,double r1, double r2, double r, double *cuda_u_old, double *cuda_u_new, double *cuda_b, double *cuda_resid, int M, int N);
extern "C" void cuda_twosolve(int epoch, double eps,double bound[][2], double *u_old, double *u_new, double *b, double *resid, int M, int N);
// c++ language use c ,must extern "C"
__global__
void cuda_copy(double *cuda_u_new, double *cuda_u_old, int M, int N){
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    if(j < N + 1 && i < M + 1){
        cuda_u_old[j*(M + 1) + i] = cuda_u_new[j*(M + 1) + i];
    }
}
__global__
void cuda_jacobi(double r1,double r2,double r, double *cuda_u_old, double *cuda_u_new, double *cuda_b, double *cuda_resid, int M, int N){
    
    double temp;
    
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    if (j < N + 1 && i < M + 1){
        if(j == 0 || j == N || i == 0 || i == M){
            cuda_u_new[j*(M + 1) + i] = cuda_b[j*(M + 1) + i];
        }
        else {
            temp = cuda_b[j*(M + 1) + i] + \
            r1*(cuda_u_old[j*(M + 1) + i - 1] + cuda_u_old[j*(M + 1) + i + 1]) + \
            r2*(cuda_u_old[(j - 1)*(M + 1) + i] + cuda_u_old[(j + 1)*(M + 1) + i]) - \
            r*cuda_u_old[j*(M + 1) + i];
            cuda_u_new[j*(M + 1) + i] = cuda_u_old[j*(M + 1) + i] + temp/r;
            cuda_resid[j*(M + 1) + i] = temp;
        }
    } 
}
void cuda_jacobi_solve(int epoch, double eps,double bound[][2], double *u_old, double *u_new, double *b, double *resid, int M, int N){
    double dx = (bound[0][1] - bound[0][0])/M;
    double dy = (bound[1][1] - bound[1][0])/N;
    double r1 = dy/dx, r2 = dx/dy;
    double r = 2*(r1 + r2);

    int size = (M + 1)*(N + 1)*sizeof(double);
    double *cuda_b, *cuda_u_old, *cuda_u_new, *cuda_resid;
    hipEvent_t start,stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //-------------------
    hipMalloc((void **) &cuda_b, size);
    hipMalloc((void **) &cuda_u_old, size);
    hipMalloc((void **) &cuda_u_new, size);
    hipMalloc((void **) &cuda_resid, size);

    hipMemcpy(cuda_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_u_old, u_old, size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_u_new, u_new, size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_resid, resid, size, hipMemcpyHostToDevice);

    dim3 grid_dim(ceil((M + 1)/(double)(BLOCK_DIM)),ceil((N + 1)/(double)(BLOCK_DIM)));
    dim3 block_dim(BLOCK_DIM, BLOCK_DIM);

    int k = 0;
    
    hipEventRecord(start,0);
    while (k < epoch){
        
        cuda_jacobi<<<grid_dim, block_dim>>>(r1,r2,r, cuda_u_old, cuda_u_new, cuda_b, cuda_resid, M, N);
        hipDeviceSynchronize();// must wait
        
        hipError_t error = hipGetLastError();
        if (error != hipSuccess && k == 0) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
    // 处理CUDA错误
        }
        
        cuda_copy<<<grid_dim, block_dim>>>(cuda_u_new, cuda_u_old, M, N);
        
        k += 1;
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop);// must float ker_time
    double res = cuda_norm(cuda_resid, M, N);
    hipMemcpy(b, cuda_b, size, hipMemcpyDeviceToHost);
    hipMemcpy(u_old, cuda_u_old, size, hipMemcpyDeviceToHost);
    hipMemcpy(u_new, cuda_u_new, size, hipMemcpyDeviceToHost);
    hipMemcpy(resid, cuda_resid, size, hipMemcpyDeviceToHost);

    hipFree(cuda_b);
    hipFree(cuda_u_old);
    hipFree(cuda_u_new);
    hipFree(cuda_resid);
    printf("grid dim: %d, %d\n",grid_dim.x, grid_dim.y);
    printf("block dim: %d, %d\n",block_dim.x, block_dim.y);
    printf("kernel launch time:%.5f,res:%.3e\n",ker_time/1000.,res);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    //----------------
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s \n",hipGetErrorString(err));
    }

    size_t free, total;
    hipMemGetInfo(&free,&total);
    printf("Free memory: %zu\n Total memory: %zu\n", free, total);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,0);
    printf("Max threads per block: %d \n Max grid size: %d,%d,%d \n",
    prop.maxThreadsPerBlock,prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
    /***
    cudaFuncAttributes attr;
    cudaFuncGetAttributes(&attr,myKernel);
    printf("Max dynamic shared memory size: %zu\n",attrsharedSizeBytes);
    ***/
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Error: %s \n", hipGetErrorString(err));
    }
}
//----------------------------------------------------------------
__global__
void cuda_init0(double *cuda_u, int M, int N){
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    if(j < N + 1 && i < M + 1){
        cuda_u[j*(M + 1) + i] = 0;
    }
}
__global__
void cuda_low(int M, int N, double *cuda_u_long, double *cuda_u_short){
    int m = M/2;
    int n = N/2;
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    //----------------
    cuda_u_short[0] = cuda_u_long[0];
    cuda_u_short[m] = cuda_u_long[M];
    cuda_u_short[n*(m + 1)] = cuda_u_long[N*(M + 1)];
    cuda_u_short[n*(m + 1) + m] = cuda_u_long[N*(M + 1) + M];
    double temp = 0;
    //--------------------
    if (j > 0 && j < n){
        cuda_u_short[j*(m + 1)] = 0.5*(cuda_u_long[(2*j - 1)*(M + 1)] + cuda_u_long[(2*j + 1)*(M + 1)]);
        cuda_u_short[j*(m + 1) + m] = 0.5*(cuda_u_long[(2*j - 1)*(M + 1) + M] + cuda_u_long[(2*j + 1)*(M + 1) + M]);
    }
    else if(i > 0 && i < m){
        cuda_u_short[i] = 0.5*(cuda_u_long[2*i - 1] + cuda_u_long[2*i + 1]);
        cuda_u_short[n*(m + 1) + i] = 0.5*(cuda_u_long[N*(M + 1) + 2*i - 1] + cuda_u_long[N*(M + 1) + 2*i + 1]);
    }
    else if(j > 0 && j < n && i > 0 && i < m){
        temp = cuda_u_long[(2*j - 1)*(M + 1) + 2*i - 1] + cuda_u_long[(2*j - 1)*(M + 1) + 2*i + 1] + \
            cuda_u_long[(2*j + 1)*(M + 1) + 2*i - 1] + cuda_u_long[(2*j + 1)*(M + 1) + 2*i + 1] + \
            2*(cuda_u_long[(2*j - 1)*(M + 1) + 2*i] + cuda_u_long[(2*j + 1)*(M + 1) + 2*i]) + \
            2*(cuda_u_long[2*j*(M + 1) + 2*i - 1] + cuda_u_long[2*j*(M + 1) + 2*i + 1]) + \
            4*cuda_u_long[2*j*(M + 1) + 2*i];
        cuda_u_short[j*(m + 1) + i] = temp/16.0;
    }
}
__global__
void cuda_high(int m, int n, double *cuda_u_short, double *cuda_u_long){
    int M = 2*m;
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    if(j < n && i < m){
        cuda_u_long[2*j*(M + 1) + 2*i] = cuda_u_short[j*(m + 1) + i];
        cuda_u_long[(2*j + 1)*(M + 1) + 2*i] = 0.5*(cuda_u_short[j*(m + 1) + i] + cuda_u_short[(j + 1)*(m + 1) + i]);
        cuda_u_long[2*j*(M + 1) + 2*i + 1] = 0.5*(cuda_u_short[j*(m + 1) + i] + cuda_u_short[j*(m + 1) + i + 1]);
        cuda_u_long[(2*j + 1)*(M + 1) + 2*i + 1] = 0.25*\
            (cuda_u_short[(j + 1)*(m + 1) + i] + cuda_u_short[(j + 1)*(m + 1) + i + 1] + cuda_u_short[j*(m + 1) + i + 1] + cuda_u_short[j*(m + 1) + i]);
    }
}
__global__
void cuda_plus(double *cuda_y, double *cuda_x, double a, int M, int N){
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    if(j < N + 1 && i < M + 1){
        cuda_y[j*(M + 1) + i] = cuda_y[j*(M + 1) + i] + a*cuda_x[j*(M + 1) + i];
    }
}
double cuda_norm(double *cuda_u, int M, int N){
    double f = 0;
    int size = (M + 1)*(N + 1)*sizeof(double);
    double *host_u;
    host_u = (double *)malloc(size);
    hipMemcpy(host_u, cuda_u, size, hipMemcpyDeviceToHost);
    for(int i = 0; i < (M + 1)*(N + 1); i++){
        f += pow(host_u[i], 2);
    }
    return sqrt(f/((M + 1)*(N + 1)));
}
void device_jacobi(int epoch, double eps,double r1, double r2, double r, double *cuda_u_old, double *cuda_u_new, double *cuda_b, double *cuda_resid, int M, int N){
    int k = 0;
    dim3 grid_dim(ceil((M + 1)/(double)(BLOCK_DIM)),ceil((N + 1)/(double)(BLOCK_DIM)));
    dim3 block_dim(BLOCK_DIM, BLOCK_DIM);
    while (k < epoch){
        cuda_jacobi<<<grid_dim, block_dim>>>(r1,r2,r, cuda_u_old, cuda_u_new, cuda_b, cuda_resid, M, N);
        hipDeviceSynchronize();// must wait
        cuda_copy<<<grid_dim, block_dim>>>(cuda_u_new, cuda_u_old, M, N);
        k += 1;
    }
}
void cuda_twosolve(int epoch, double eps,double bound[][2], double *u_old, double *u_new, double *b, double *resid, int M, int N){
    double dx = (bound[0][1] - bound[0][0])/M;
    double dy = (bound[1][1] - bound[1][0])/N;
    double r1 = dy/dx, r2 = dx/dy;
    double r = 2*(r1 + r2);
    int gepoch = 20;
    hipEvent_t start,stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //-------------------
    int cuda_size = (M + 1)*(N + 1)*sizeof(double);
    double *cuda_u_old, *cuda_u_new, *cuda_b, *cuda_resid;

    hipMalloc((void **) &cuda_b, cuda_size);
    hipMalloc((void **) &cuda_resid, cuda_size);
    hipMalloc((void **) &cuda_u_old, cuda_size);
    hipMalloc((void **) &cuda_u_new, cuda_size);

    hipMemcpy(cuda_b, b, cuda_size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_resid, resid, cuda_size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_u_old, u_old, cuda_size, hipMemcpyHostToDevice);
    hipMemcpy(cuda_u_new, u_new, cuda_size, hipMemcpyHostToDevice);
    //-------------------
    int m = M/2,n = N/2;
    int sizesmall = (m + 1)*(n + 1)*sizeof(double);
    double *cuda_b_short, *cuda_u_short, *cuda_u_old_short, *cuda_resid_short;

    hipMalloc((void **) &cuda_b_short, sizesmall);
    hipMalloc((void **) &cuda_resid_short, sizesmall);
    hipMalloc((void **) &cuda_u_short, sizesmall);
    hipMalloc((void **) &cuda_u_old_short, sizesmall);

    dim3 grid_dim(ceil((M + 1)/(double)(BLOCK_DIM)),ceil((N + 1)/(double)(BLOCK_DIM)));
    dim3 block_dim(BLOCK_DIM, BLOCK_DIM);
    double norm = 0;
    hipEventRecord(start,0);
    int k = 0;
    while(k < epoch){
        device_jacobi(gepoch, eps, r1,r2,r, cuda_u_old, cuda_u_new, cuda_b, cuda_resid, M, N);
        norm = cuda_norm(cuda_u_new, M, N);
        cuda_low<<<grid_dim, block_dim>>>(M, N, cuda_resid, cuda_b_short);
        hipDeviceSynchronize();// must wait
        cuda_init0<<<grid_dim, block_dim>>>(cuda_u_old_short, m, n);
        hipDeviceSynchronize();// must wait
        device_jacobi(gepoch, eps, r1,r2,r, cuda_u_old_short, cuda_u_short, cuda_b_short, cuda_resid_short, m,n);
        cuda_init0<<<grid_dim, block_dim>>>(cuda_u_old,M,N);
        hipDeviceSynchronize();// must wait
        cuda_high<<<grid_dim, block_dim>>>(m, n, cuda_u_short, cuda_u_old);
        hipDeviceSynchronize();// must wait
        cuda_plus<<<grid_dim, block_dim>>>(cuda_u_new, cuda_u_old, 1.0, M, N);
        hipDeviceSynchronize();// must wait
        cuda_copy<<<grid_dim, block_dim>>>(cuda_u_new, cuda_u_old, M, N);
        hipDeviceSynchronize();// must wait
        k += 1;
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop);// must float ker_time
    norm = cuda_norm(cuda_resid, M, N);
    hipMemcpy(b, cuda_b, cuda_size, hipMemcpyDeviceToHost);
    hipMemcpy(resid, cuda_resid, cuda_size, hipMemcpyDeviceToHost);
    hipMemcpy(u_old, cuda_u_old, cuda_size, hipMemcpyDeviceToHost);
    hipMemcpy(u_new, cuda_u_new, cuda_size, hipMemcpyDeviceToHost);

    hipFree(cuda_b);
    hipFree(cuda_resid);
    hipFree(cuda_u_old);
    hipFree(cuda_u_new);
    hipFree(cuda_resid_short);
    hipFree(cuda_b_short);
    hipFree(cuda_u_old_short);
    hipFree(cuda_u_short);
    printf("grid dim: %d, %d\n",grid_dim.x, grid_dim.y);
    printf("block dim: %d, %d\n",block_dim.x, block_dim.y);
    printf("kernel launch time:%.5f,resid:%.3e\n",ker_time/1000.,norm);

    hipEventDestroy(start);
    hipEventDestroy(stop);

}

