#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cub/block/block_reduce.cuh>
#include <>

#define max_function(a, b) ((a) > (b) ? (a) : (b))

double get_walltime() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return (double) (tp.tv_sec + tp.tv_usec*1e-6); 
}

template <int BLOCK_DIM_y>
__launch_bounds__(BLOCK_DIM_y) __global__
    void _attentionKernel(const float *__restrict inputQ,
                          const float *__restrict inputK,
                          const float *__restrict inputV, int N, int d,
                          float *__restrict output) {
    int i = blockIdx.x;                              // i must < N,Q[i]
    int phd = threadIdx.y + blockIdx.y * blockDim.y; // V[:,d]

    __shared__ float old_max[BLOCK_DIM_y];
    __shared__ float new_max[BLOCK_DIM_y];
    __shared__ float new_sum[BLOCK_DIM_y];
    old_max[threadIdx.y] = -__FLT_MAX__;
    new_max[threadIdx.y] = -__FLT_MAX__;
    new_sum[threadIdx.y] = 0.0f;
    __shared__ float block_sum[BLOCK_DIM_y];
    __shared__ float block_max[BLOCK_DIM_y];

    __shared__ float inputS[BLOCK_DIM_y];
    __shared__ float shareV[BLOCK_DIM_y];
    __shared__ float out[BLOCK_DIM_y];

    int phNumD = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
    __shared__ float shareQ_times_K[BLOCK_DIM_y];
    
    for (int phn = 0; phn < N; phn++) {
        shareV[threadIdx.y] = 0.0f;

        float sum_s = 0.0f;
        for (int ind = 0; ind < phNumD; ind++) {
            if (threadIdx.y + ind * BLOCK_DIM_y < d) {
                shareQ_times_K[threadIdx.y] =
                    inputQ[i * d + threadIdx.y + ind * BLOCK_DIM_y] * inputK[phn * d + threadIdx.y + ind * BLOCK_DIM_y];
                
            } else {
                shareQ_times_K[threadIdx.y] = 0.0f;
                
            }
            __syncthreads();
            for(int strip = BLOCK_DIM_y/2; strip > 0; strip = strip/2){
                if(threadIdx.y < strip){
                    shareQ_times_K[threadIdx.y] += shareQ_times_K[threadIdx.y + strip];
                }
                __syncthreads();
            }
            sum_s += shareQ_times_K[0];
            __syncthreads();
        }

        inputS[threadIdx.y] = sum_s;
        block_max[threadIdx.y] = sum_s;
        block_sum[threadIdx.y] = 1.0f;

        if (phd < d) {
            shareV[threadIdx.y] = inputV[phn * d + phd];
        }

        __syncthreads();

        if (new_max[threadIdx.y] > block_max[threadIdx.y]) {
            new_sum[threadIdx.y] =
                new_sum[threadIdx.y] +
                block_sum[threadIdx.y] *
                    __expf(block_max[threadIdx.y] - new_max[threadIdx.y]);
        } else {
            new_sum[threadIdx.y] =
                block_sum[threadIdx.y] +
                new_sum[threadIdx.y] *
                    __expf(new_max[threadIdx.y] - block_max[threadIdx.y]);
            new_max[threadIdx.y] = block_max[threadIdx.y];
        }

        __syncthreads();

        inputS[threadIdx.y] =
            __expf(inputS[threadIdx.y] - new_max[threadIdx.y]);

        __syncthreads();

        if (phn == 0) {
            out[threadIdx.y] = inputS[threadIdx.y] * shareV[threadIdx.y];

        } else {
            out[threadIdx.y] =
                __expf(old_max[threadIdx.y] - new_max[threadIdx.y]) *
                    out[threadIdx.y] +
                inputS[threadIdx.y] * shareV[threadIdx.y];
        }

        old_max[threadIdx.y] = new_max[threadIdx.y];

        __syncthreads();
    }
    __syncthreads();
    if (phd < d)
        output[i * d + phd] =
            out[threadIdx.y] * __fdividef(1.0F, new_sum[threadIdx.y]);
}
void attention(float *cpu_Q, float *cpu_K, float *cpu_V, int N, int d, float *cpu_output){
    double st, ela;
    st = get_walltime();
    
    float *inputQ, *inputK, *inputV, *output;
    hipMalloc((void **) &inputQ, N*d*sizeof(float));
    hipMalloc((void **) &inputK, N*d*sizeof(float));
    hipMalloc((void **) &inputV, N*d*sizeof(float));
    
    hipMalloc((void **) &output, N*d*sizeof(float));
    hipMemcpy(inputQ, cpu_Q, N*d*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputK, cpu_K, N*d*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputV, cpu_V, N*d*sizeof(float), hipMemcpyHostToDevice);
    
    
    hipEvent_t start,stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    int num_block_x = N;

    if (d > 1023) {
        int BLOCK_DIM_y = 1024;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<1024>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 511) {
        int BLOCK_DIM_y = 512;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<512>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 255) {
        int BLOCK_DIM_y = 256;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<256>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 127) {
        int BLOCK_DIM_y = 128;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<128>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 63) {
        int BLOCK_DIM_y = 64;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<64>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 31) {
        int BLOCK_DIM_y = 32;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<32>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else {
        int BLOCK_DIM_y = 16;
        int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
        dim3 block_dim(1, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<16>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    }
        
        
    
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop);// must float ker_time
    hipMemcpy(cpu_output, output, N*d*sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(inputQ);
    hipFree(inputK);
    hipFree(inputV);
    
    hipFree(output);
    
    ela = get_walltime() - st;
    
    printf("BlockReduce,kernel time:%.4f, use time:%.4f\n", ker_time/1000., ela);
    
}
int main() {
    int N = 1024;
    int d = 1000;
    
    int size = N*d;
    
    
    float *cpu_Q, *cpu_K, *cpu_V, *cpu_output;
    cpu_Q = (float *)malloc(size*sizeof(float));
    cpu_K = (float *)malloc(size*sizeof(float));
    cpu_V = (float *)malloc(size*sizeof(float));
    cpu_output = (float *)malloc(size*sizeof(float));
    for(int i = 0; i < size; i++){
        cpu_Q[i] = i%4;
        cpu_K[i] = i%4;
        cpu_V[i] = i%4;
        //printf("Q:%.4f\n",cpu_Q[i]);
    }
    
    
    attention(cpu_Q, cpu_K, cpu_V, N, d, cpu_output);
    for(int i = 0; i < 10; i++){
        
        printf("out:%.6e\n",cpu_output[i]);
    }
    
    
    free(cpu_Q);
    free(cpu_K);
    free(cpu_V);
    free(cpu_output);
    
    return 0;
}






