#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cub/block/block_reduce.cuh>
#include <>

double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}

__global__ void _attentionKernel(const float *__restrict inputQ,
                                 const float *__restrict inputK,
                                 const float *__restrict inputV, int N, int d, int Br, int Bc,
                                 float *__restrict output)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; // i must < N,Q[i]
    int phd = blockIdx.y;                          // V[:,d]

    int Tc = (N + Bc - 1) / Bc;
    float newMax;
    float oldMax;
    float newSum;

    newMax = -__FLT_MAX__;
    oldMax = -__FLT_MAX__;
    newSum = 0.0f;

    float out;
    out = 0.0f;
    //---------
    extern __shared__ float sram[];
    float *block_sum = sram;
    float *block_max = sram + Br * Bc;
    float *sumSV = sram + Br * Bc * 2;

    for (int phn = 0; phn < Tc; phn++)
    {
        int j = threadIdx.y + phn * Bc;
        float sum_s = 0.0f;
        for (int index = 0; index < d; index++)
        {
            sum_s += inputQ[i * d + index] * inputK[j * d + index];
        }

        if (i < N && j < N)
        {

            block_max[threadIdx.x * Bc + threadIdx.y] = sum_s;
            block_sum[threadIdx.x * Bc + threadIdx.y] = 1.0f;
        }
        else
        {

            block_max[threadIdx.x * Bc + threadIdx.y] = -__FLT_MAX__;
            block_sum[threadIdx.x * Bc + threadIdx.y] = 0.0f;
        }
        __syncthreads();
        for (int strip = Bc / 2; strip > 0; strip /= 2)
        {
            if (threadIdx.y < strip)
            {
                if (block_max[threadIdx.x * Bc + threadIdx.y] >
                    block_max[threadIdx.x * Bc + threadIdx.y + strip])
                {
                    block_sum[threadIdx.x * Bc + threadIdx.y] =
                        block_sum[threadIdx.x * Bc + threadIdx.y] +
                        block_sum[threadIdx.x * Bc + threadIdx.y + strip] *
                            __expf(block_max[threadIdx.x * Bc + threadIdx.y + strip] -
                                   block_max[threadIdx.x * Bc + threadIdx.y]);
                }
                else
                {
                    block_sum[threadIdx.x * Bc + threadIdx.y] =
                        block_sum[threadIdx.x * Bc + threadIdx.y + strip] +
                        block_sum[threadIdx.x * Bc + threadIdx.y] *
                            __expf(block_max[threadIdx.x * Bc + threadIdx.y] -
                                   block_max[threadIdx.x * Bc + threadIdx.y + strip]);
                    block_max[threadIdx.x * Bc + threadIdx.y] =
                        block_max[threadIdx.x * Bc + threadIdx.y + strip];
                }
            }
            __syncthreads();
        }
        if (newMax > block_max[threadIdx.x * Bc])
        {
            newSum = newSum + block_sum[threadIdx.x * Bc] *
                                  __expf(block_max[threadIdx.x * Bc] - newMax);
        }
        else
        {
            newSum = block_sum[threadIdx.x * Bc] +
                     newSum * __expf(newMax - block_max[threadIdx.x * Bc]);
            newMax = block_max[threadIdx.x * Bc];
        }

        __syncthreads();
        if (i < N && j < N)
        {
            sumSV[threadIdx.x * Bc + threadIdx.y] =
                __expf(sum_s - newMax) *
                inputV[(threadIdx.y + phn * Bc) * d + phd];
        }
        else
        {
            sumSV[threadIdx.x * Bc + threadIdx.y] = 0.0f;
        }
        __syncthreads();
        for (int strip = Bc / 2; strip > 0; strip /= 2)
        {
            if (threadIdx.y < strip)
            {
                sumSV[threadIdx.x * Bc + threadIdx.y] +=
                    sumSV[threadIdx.x * Bc + threadIdx.y + strip];
            }
            __syncthreads();
        }
        if (i < N && j < N)
        {
            out = __expf(oldMax - newMax) * out + sumSV[threadIdx.x * Bc];
        }
        oldMax = newMax;
        __syncthreads();
    }

    if (threadIdx.y + (Tc - 1) * Bc < N && i < N)
    {
        output[i * d + phd] = out * __fdividef(1.0F, newSum);
    }
}
void attention(float *cpu_Q, float *cpu_K, float *cpu_V, int N, int d, float *cpu_output)
{
    double st, ela;
    st = get_walltime();

    float *inputQ, *inputK, *inputV, *output;
    hipMalloc((void **)&inputQ, N * d * sizeof(float));
    hipMalloc((void **)&inputK, N * d * sizeof(float));
    hipMalloc((void **)&inputV, N * d * sizeof(float));

    hipMalloc((void **)&output, N * d * sizeof(float));
    hipMemcpy(inputQ, cpu_Q, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputK, cpu_K, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputV, cpu_V, N * d * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    int num_block_y = d;
    int Br = 32;
    int Bc = 32;
    int num_block_x = (N + Br - 1) / Br;
    dim3 block_dim(Br, Bc, 1);
    dim3 grid_dim(num_block_x, num_block_y, 1);
    int share_mem = 3 * Br * Bc * sizeof(float); // 由于global函数里面未明确分配内存，此时必须指定共享内存分配大小
    _attentionKernel<<<grid_dim, block_dim, share_mem>>>(inputQ, inputK, inputV, N, d, Br, Bc, output);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop); // must float ker_time
    hipMemcpy(cpu_output, output, N * d * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(inputQ);
    hipFree(inputK);
    hipFree(inputV);

    hipFree(output);

    ela = get_walltime() - st;

    printf("BlockReduce,kernel time:%.4f, use time:%.4f\n", ker_time / 1000., ela);
}
int main()
{
    int N = 4;
    int d = 3;

    int size = N * d;

    float *cpu_Q, *cpu_K, *cpu_V, *cpu_output;
    cpu_Q = (float *)malloc(size * sizeof(float));
    cpu_K = (float *)malloc(size * sizeof(float));
    cpu_V = (float *)malloc(size * sizeof(float));
    cpu_output = (float *)malloc(size * sizeof(float));
    for (int i = 0; i < size; i++)
    {
        cpu_Q[i] = i % 4;
        cpu_K[i] = i % 4;
        cpu_V[i] = i % 4;
        // printf("Q:%.4f\n",cpu_Q[i]);
    }

    attention(cpu_Q, cpu_K, cpu_V, N, d, cpu_output);
    for (int i = 0; i < 10; i++)
    {

        printf("out:%.6e\n", cpu_output[i]);
    }

    free(cpu_Q);
    free(cpu_K);
    free(cpu_V);
    free(cpu_output);

    return 0;
}


