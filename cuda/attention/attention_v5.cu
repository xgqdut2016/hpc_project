#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cub/block/block_reduce.cuh>
#include <>



double get_walltime() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return (double) (tp.tv_sec + tp.tv_usec*1e-6); 
}
template <int BLOCK_DIM_x, int BLOCK_DIM_y>
__global__ void _attentionKernel(const float *__restrict inputQ,
                                 const float *__restrict inputK,
                                 const float *__restrict inputV, int N, int d,
                                 float *__restrict output) {
    int i = blockIdx.y;                              // i must < N,Q[i]
    int phd = threadIdx.x + blockIdx.x * blockDim.x; // V[:,d]

    int phNumN = (N + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
    __shared__ float inputS[BLOCK_DIM_x][BLOCK_DIM_y];
    float newMax;
    float oldMax;
    float newSum;

    newMax = -__FLT_MAX__;
    oldMax = -__FLT_MAX__;
    newSum = 0.0f;

    float out;
    out = 0.0f;
    //---------
    __shared__ float block_sum[BLOCK_DIM_x][BLOCK_DIM_y];
    __shared__ float block_max[BLOCK_DIM_x][BLOCK_DIM_y];

    for (int phn = 0; phn < phNumN; phn++) {

        int j = threadIdx.y + phn * BLOCK_DIM_y;
        float sum_s = 0.0f;
        for (int index = 0; index < d; index++) {
            sum_s += inputQ[i * d + index] * inputK[j * d + index];
        }

        if (j < N) {

            block_max[threadIdx.x][threadIdx.y] = sum_s;
            block_sum[threadIdx.x][threadIdx.y] = 1.0f;
        } else {

            block_max[threadIdx.x][threadIdx.y] = -__FLT_MAX__;
            block_sum[threadIdx.x][threadIdx.y] = 0.0f;
        }
        __syncthreads();
        for (int strip = BLOCK_DIM_y / 2; strip > 0; strip /= 2) {
            if (threadIdx.y < strip) {
                if (block_max[threadIdx.x][threadIdx.y] >
                    block_max[threadIdx.x][threadIdx.y + strip]) {
                    block_sum[threadIdx.x][threadIdx.y] =
                        block_sum[threadIdx.x][threadIdx.y] +
                        block_sum[threadIdx.x][threadIdx.y + strip] *
                            __expf(block_max[threadIdx.x][threadIdx.y + strip] -
                                   block_max[threadIdx.x][threadIdx.y]);
                } else {
                    block_sum[threadIdx.x][threadIdx.y] =
                        block_sum[threadIdx.x][threadIdx.y + strip] +
                        block_sum[threadIdx.x][threadIdx.y] *
                            __expf(block_max[threadIdx.x][threadIdx.y] -
                                   block_max[threadIdx.x][threadIdx.y + strip]);
                    block_max[threadIdx.x][threadIdx.y] =
                        block_max[threadIdx.x][threadIdx.y + strip];
                }
            }
            __syncthreads();
        }
        if (newMax > block_max[threadIdx.x][0]) {
            newSum = newSum + block_sum[threadIdx.x][0] *
                                  __expf(block_max[threadIdx.x][0] - newMax);
        } else {
            newSum = block_sum[threadIdx.x][0] +
                     newSum * __expf(newMax - block_max[threadIdx.x][0]);
            newMax = block_max[threadIdx.x][0];
        }

        __syncthreads();
        if (j < N && phd < d) {
            inputS[threadIdx.x][threadIdx.y] = __expf(sum_s - newMax);
        } else {
            inputS[threadIdx.x][threadIdx.y] = 0.0f;
        }
        __syncthreads();
        float sum_o = 0.0f;
        for (int index = 0; index < BLOCK_DIM_y; index++) {
            sum_o += inputS[threadIdx.x][index] *
                     inputV[(index + phn * BLOCK_DIM_y) * d + phd];
        }
        if (j < N && phd < d) {
            out = __expf(oldMax - newMax) * out + sum_o;
        }
        oldMax = newMax;
        __syncthreads();
    }

    if (threadIdx.y + (phNumN - 1) * BLOCK_DIM_y < N && phd < d) {
        output[i * d + phd] = out * __fdividef(1.0F, newSum);
    }
}
void attention(float *cpu_Q, float *cpu_K, float *cpu_V, int N, int d, float *cpu_output){
    double st, ela;
    st = get_walltime();
    
    float *inputQ, *inputK, *inputV, *output;
    hipMalloc((void **) &inputQ, N*d*sizeof(float));
    hipMalloc((void **) &inputK, N*d*sizeof(float));
    hipMalloc((void **) &inputV, N*d*sizeof(float));
    
    
    hipMalloc((void **) &output, N*d*sizeof(float));
    hipMemcpy(inputQ, cpu_Q, N*d*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputK, cpu_K, N*d*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputV, cpu_V, N*d*sizeof(float), hipMemcpyHostToDevice);
    
    
    hipEvent_t start,stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    int num_block_y = N;
    if (d > 512) {
        int BLOCK_DIM_x = 1024;
        int BLOCK_DIM_y = 1;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<1024, 1>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 256) {
        int BLOCK_DIM_x = 512;
        int BLOCK_DIM_y = 2;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<512, 2>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 128) {
        int BLOCK_DIM_x = 256;
        int BLOCK_DIM_y = 4;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<256, 4>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 64) {
        int BLOCK_DIM_x = 128;
        int BLOCK_DIM_y = 8;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<128, 8>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 32) {
        int BLOCK_DIM_x = 64;
        int BLOCK_DIM_y = 16;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<64, 16>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else if (d > 16) {
        int BLOCK_DIM_x = 32;
        int BLOCK_DIM_y = 32;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<32, 32>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    } else {
        int BLOCK_DIM_x = 16;
        int BLOCK_DIM_y = 64;
        int num_block_x = (d + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
        dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
        dim3 grid_dim(num_block_x, num_block_y, 1);
        _attentionKernel<16, 64>
            <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    }
        
        
    
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop);// must float ker_time
    hipMemcpy(cpu_output, output, N*d*sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(inputQ);
    hipFree(inputK);
    hipFree(inputV);
    
    hipFree(output);
    
    ela = get_walltime() - st;
    
    printf("BlockReduce,kernel time:%.4f, use time:%.4f\n", ker_time/1000., ela);
    
}
int main() {
    int N = 4;
    int d = 3;
    
    int size = N*d;
    
    
    float *cpu_Q, *cpu_K, *cpu_V, *cpu_output;
    cpu_Q = (float *)malloc(size*sizeof(float));
    cpu_K = (float *)malloc(size*sizeof(float));
    cpu_V = (float *)malloc(size*sizeof(float));
    cpu_output = (float *)malloc(size*sizeof(float));
    for(int i = 0; i < size; i++){
        cpu_Q[i] = i%4;
        cpu_K[i] = i%4;
        cpu_V[i] = i%4;
        //printf("Q:%.4f\n",cpu_Q[i]);
    }
    
    
    attention(cpu_Q, cpu_K, cpu_V, N, d, cpu_output);
    for(int i = 0; i < 10; i++){
        
        printf("out:%.6e\n",cpu_output[i]);
    }
    
    
    free(cpu_Q);
    free(cpu_K);
    free(cpu_V);
    free(cpu_output);
    
    return 0;
}







