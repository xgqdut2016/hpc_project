#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cub/block/block_reduce.cuh>
#include <>



double get_walltime() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return (double) (tp.tv_sec + tp.tv_usec*1e-6); 
}
#define BLOCK_DIM_x 32
#define BLOCK_DIM_y 32
#define max_function(a, b) ((a) > (b) ? (a) : (b))

__global__ void _attentionKernel(const float *inputQ, const float *inputK,
                                 const float *inputV, int N, int d,
                                 float *output) {
    int i = blockIdx.x;                              // i must < N,Q[i]
    int phd = threadIdx.y + blockIdx.y * blockDim.y; // V[:,d]
    int phNumN = (N + BLOCK_DIM_x - 1) / BLOCK_DIM_x;
    __shared__ float old_max[BLOCK_DIM_x][BLOCK_DIM_y];
    __shared__ float new_max[BLOCK_DIM_x][BLOCK_DIM_y];
    __shared__ float new_sum[BLOCK_DIM_x][BLOCK_DIM_y];
    old_max[threadIdx.x][threadIdx.y] = -__FLT_MAX__;
    new_max[threadIdx.x][threadIdx.y] = -__FLT_MAX__;
    new_sum[threadIdx.x][threadIdx.y] = 0.0f;
    __shared__ float block_sum[BLOCK_DIM_x][BLOCK_DIM_y];
    __shared__ float block_max[BLOCK_DIM_x][BLOCK_DIM_y];
    
    __shared__ float inputS[BLOCK_DIM_x][BLOCK_DIM_y];
    __shared__ float shareV[BLOCK_DIM_x][BLOCK_DIM_y];
    __shared__ float out[BLOCK_DIM_y];
    for (int phn = 0; phn < phNumN; phn++) {
        int j = threadIdx.x + phn * BLOCK_DIM_x;
        inputS[threadIdx.x][threadIdx.y] = 0.0f;
        block_max[threadIdx.x][threadIdx.y] = -__FLT_MAX__;
        block_sum[threadIdx.x][threadIdx.y] = 0.0f;
        shareV[threadIdx.x][threadIdx.y] = 0.0f;

        if (j < N && phd < d) {
            float sum_s = 0;
            for(int index = 0; index < d; index++){
                sum_s += inputQ[i * d + index] * inputK[j * d + index];
            }
            shareV[threadIdx.x][threadIdx.y] = inputV[j * d + phd];
            inputS[threadIdx.x][threadIdx.y] = sum_s;
            block_max[threadIdx.x][threadIdx.y] = sum_s;
            block_sum[threadIdx.x][threadIdx.y] = 1.0f;
        }

        __syncthreads();
        for (int strip = BLOCK_DIM_x / 2; strip > 0; strip = strip / 2) {
            if (threadIdx.x < strip) {
                if (block_max[threadIdx.x][threadIdx.y] >
                    block_max[threadIdx.x + strip][threadIdx.y]) {
                    block_sum[threadIdx.x][threadIdx.y] =
                        block_sum[threadIdx.x][threadIdx.y] +
                        block_sum[threadIdx.x + strip][threadIdx.y] *
                            __expf(block_max[threadIdx.x + strip][threadIdx.y] -
                                   block_max[threadIdx.x][threadIdx.y]);
                } else {
                    block_sum[threadIdx.x][threadIdx.y] =
                        block_sum[threadIdx.x + strip][threadIdx.y] +
                        block_sum[threadIdx.x][threadIdx.y] *
                            __expf(block_max[threadIdx.x][threadIdx.y] -
                                   block_max[threadIdx.x + strip][threadIdx.y]);
                    block_max[threadIdx.x][threadIdx.y] =
                        block_max[threadIdx.x + strip][threadIdx.y];
                }
            }
            __syncthreads();
        }
        __syncthreads();
        if (j < N) {
            if (new_max[threadIdx.x][threadIdx.y] > block_max[0][threadIdx.y]) {
                new_sum[threadIdx.x][threadIdx.y] =
                    new_sum[threadIdx.x][threadIdx.y] +
                    block_sum[0][threadIdx.y] *
                        __expf(block_max[0][threadIdx.y] -
                               new_max[threadIdx.x][threadIdx.y]);
            } else {
                new_sum[threadIdx.x][threadIdx.y] =
                    block_sum[0][threadIdx.y] +
                    new_sum[threadIdx.x][threadIdx.y] *
                        __expf(new_max[threadIdx.x][threadIdx.y] -
                               block_max[0][threadIdx.y]);
                new_max[threadIdx.x][threadIdx.y] = block_max[0][threadIdx.y];
            }
        }

        __syncthreads();

        if (j < N) {
            inputS[threadIdx.x][threadIdx.y] =
                __expf(inputS[threadIdx.x][threadIdx.y] -
                       new_max[threadIdx.x][threadIdx.y]);
            
        } else {
            inputS[threadIdx.x][threadIdx.y] = 0.0f;
            
        }
        __syncthreads();

        
        float sum_o = 0.0f;
        for (int index = 0; index < BLOCK_DIM_x; index++) {
                
            sum_o += inputS[index][threadIdx.y] *
                             shareV[index][threadIdx.y];
                
        }
        if (phn == 0) {
            out[threadIdx.y] = sum_o;
                //output[i * d + phd] = sum_o;
        } else {
            out[threadIdx.y] =
                    __expf(old_max[threadIdx.x][threadIdx.y] -
                           new_max[threadIdx.x][threadIdx.y]) *
                        out[threadIdx.y] +
                    sum_o;
        }

        old_max[threadIdx.x][threadIdx.y] =
                new_max[threadIdx.x][threadIdx.y];
        __syncthreads();
    }
    __syncthreads();
    if (phd < d)
        output[i * d + phd] =
            out[threadIdx.y] *
            __fdividef(1.0F, new_sum[threadIdx.x][threadIdx.y]);
}
void attention(float *cpu_Q, float *cpu_K, float *cpu_V, int N, int d, float *cpu_output){
    double st, ela;
    st = get_walltime();
    
    float *Q, *K, *V, *S, *output;
    hipMalloc((void **) &Q, N*d*sizeof(float));
    hipMalloc((void **) &K, N*d*sizeof(float));
    hipMalloc((void **) &V, N*d*sizeof(float));
    hipMalloc((void **) &S, N*N*sizeof(float));
    
    hipMalloc((void **) &output, N*d*sizeof(float));
    hipMemcpy(Q, cpu_Q, N*d*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(K, cpu_K, N*d*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(V, cpu_V, N*d*sizeof(float), hipMemcpyHostToDevice);
    
    
    hipEvent_t start,stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    int num_block_x = N;
    int num_block_y = (d + BLOCK_DIM_y - 1) / BLOCK_DIM_y;
    dim3 block_dim(BLOCK_DIM_x, BLOCK_DIM_y, 1);
    dim3 grid_dim(num_block_x, num_block_y, 1);
    //int share_mem = (7 * BLOCK_DIM_x) * BLOCK_DIM_y * sizeof(float);
    _attentionKernel<<<grid_dim, block_dim>>>(Q, K, V, N, d, output);
        
        
    
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop);// must float ker_time
    hipMemcpy(cpu_output, output, N*d*sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(Q);
    hipFree(K);
    hipFree(V);
    hipFree(S);
    hipFree(output);
    
    ela = get_walltime() - st;
    
    printf("BlockReduce,kernel time:%.4f, use time:%.4f\n", ker_time/1000., ela);
    
}
int main() {
    int N = 4;
    int d = 3;
    
    int size = N*d;
    
    
    float *cpu_Q, *cpu_K, *cpu_V, *cpu_output;
    cpu_Q = (float *)malloc(size*sizeof(float));
    cpu_K = (float *)malloc(size*sizeof(float));
    cpu_V = (float *)malloc(size*sizeof(float));
    cpu_output = (float *)malloc(size*sizeof(float));
    for(int i = 0; i < size; i++){
        cpu_Q[i] = i%4;
        cpu_K[i] = i%4;
        cpu_V[i] = i%4;
        //printf("Q:%.4f\n",cpu_Q[i]);
    }
    
    
    attention(cpu_Q, cpu_K, cpu_V, N, d, cpu_output);
    for(int i = 0; i < 10; i++){
        
        printf("out:%.6e\n",cpu_output[i]);
    }
    
    
    free(cpu_Q);
    free(cpu_K);
    free(cpu_V);
    free(cpu_output);
    
    return 0;
}






