#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cub/block/block_reduce.cuh>
#include <>


double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}

const int Rq = 4;
const int Rv = 8; // 必须是4的倍数
const int Br = 16;
const int Bc = 16;
const int Bk = 4; // 必须是4的倍数
const int Bd = 16;
const int numQ = Rq * Br;
const int numK = Bk * Bc;
const int numV = Rv * Bc;

__device__ void matmulRQK(const float *__restrict inputQ,
                          const float *__restrict inputK, float *shareQK,
                          float *shareVK, int N, int d, int width, int indQ,
                          int indK, float *val)
{
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    float a[4];
    for (int ph = 0; ph < width; ph++)
    {
        int Q_smem = (tid / 4) * Bd + (tid % 4) * 4;
        int Q_gmem = (indQ + tid / 4) * d + Bd * ph + (tid % 4) * 4;

        (float4 &)shareQK[Q_smem] = (float4 &)inputQ[Q_gmem];

        int K_gmem = (indK + tid % 64) * d + Bd * ph + (tid / 64) * 4;
        (float4 &)a[0] = (float4 &)inputK[K_gmem];
        for (int id = 0; id < 4; id++)
        {
            shareVK[((tid / 64) * 4 + id) * numK + (tid % 64)] = a[id];
        }
        __syncthreads();
        for (int index = 0; index < Bd; index++)
        {
            for (int index_q = 0; index_q < Rq; index_q++)
            {
                for (int index_k = 0; index_k < Bk; index_k++)
                {
                    int comp_a_smem_m = threadIdx.y * Rq + index_q;
                    int comp_b_smem_n = threadIdx.x * Bk + index_k;
                    val[index_q * Rq + index_k] +=
                        shareQK[comp_a_smem_m * Bd + index] *
                        shareVK[comp_b_smem_n + index * numK];
                }
            }
        }
        __syncthreads();
    }
}

__device__ void matmulSV(float *shareQK, const float *__restrict inputV,
                         float *shareVK, int N, int d, int j, int indQ,
                         int indK, int indV, float *val, float *newMax,
                         float *sumSV)
{
    if (threadIdx.y < Bc)
    {
        for (int index_k = 0; index_k < Bk; index_k++)
        {
            for (int id = 0; id < Rv; id += 4)
            {
                (float4 &)shareVK[(threadIdx.y * Bk + index_k) * Bc * Rv +
                                  threadIdx.x * Rv + id] = (float4 &)
                    inputV[((threadIdx.y + j * Bc) * Bk + index_k) * d + indV +
                           threadIdx.x * Rv + id];
            }
            for (int index_v = 0; index_v < Rv; index_v++)
            {
                if ((threadIdx.y + j * Bc) * Bk + index_k >= N ||
                    indV + threadIdx.x * Rv + index_v >= d)
                {
                    shareVK[(threadIdx.y * Bk + index_k) * Bc * Rv +
                            threadIdx.x * Rv + index_v] = 0.0f;
                }
            }
        }
    }
    for (int index_q = 0; index_q < Rq; index_q++)
    {
        for (int index_k = 0; index_k < Bk; index_k++)
        {
            if (indQ + threadIdx.y * Rq + index_q < N &&
                indK + Bk * threadIdx.x + index_k < N)
            {
                shareQK[(threadIdx.y * Rq + index_q) * numK + threadIdx.x * Bk +
                        index_k] =
                    __expf(val[index_q * Bk + index_k] - newMax[index_q]);
            }
            else
            {

                shareQK[(threadIdx.y * Rq + index_q) * numK + threadIdx.x * Bk +
                        index_k] = 0.0f;
            }
        }
    }
    __syncthreads();

    for (int phc = 0; phc < numK; phc++)
    {
        for (int index_q = 0; index_q < Rq; index_q++)
        {

            for (int index_v = 0; index_v < Rv; index_v++)
            {
                sumSV[index_q * Rv + index_v] +=
                    shareQK[(threadIdx.y * Rq + index_q) * numK + phc] *
                    shareVK[phc * Bc * Rv + threadIdx.x * Rv + index_v];
            }
        }
    }
}
template <typename T>
struct SumOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return a + b;
    }
};

template <typename T>
struct MaxOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return max(a, b);
    }
};
template <template <typename> class ReductionOp, typename T,
          int thread_group_width = 32>
__inline__ __device__ T WarpAllReduce(T val)
{
    for (int mask = thread_group_width / 2; mask > 0; mask >>= 1)
    {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }

    return val;
}

template <int Br, int Bc, int Rq, int Rv>
__global__ void _attentionKernel(const float *__restrict inputQ,
                                 const float *__restrict inputK,
                                 const float *__restrict inputV, int N, int d,
                                 float *__restrict output)
{

    __shared__ float shareQK[numQ * numK];
    __shared__ float shareVK[numK * numV];

    float sumSV[Rq * Rv] = {0.0f};
    float newMax[Rq];
    float oldMax[Rq];
    float newSum[Rq] = {0.0f};

    float val[Rq * Bk];

    int indV = Rv * blockIdx.x * blockDim.x;
    int indQ = Rq * blockIdx.y * blockDim.y;

    for (int index_q = 0; index_q < Rq; index_q++)
    {
        newMax[index_q] = -__FLT_MAX__;
        oldMax[index_q] = -__FLT_MAX__;
    }

    int Tc = (N + numK - 1) / (numK);

    int width = (d + Bd - 1) / Bd;
    for (int j = 0; j < Tc; j++)
    {

        int indK = j * numK;
        for (int index_q = 0; index_q < Rq; index_q++)
        {
            for (int index_k = 0; index_k < Bk; index_k++)
            {

                val[index_q * Bk + index_k] = 0.0f;
            }
        }
        matmulRQK(inputQ, inputK, shareQK, shareVK, N, d, width, indQ, indK,
                  val);

        for (int index_q = 0; index_q < Rq; index_q++)
        {
            float tmpReduceMax = -__FLT_MAX__;
            for (int index_k = 0; index_k < Bk; index_k++)
            {
                if (indQ + threadIdx.y * Rq + index_q < N &&
                    indK + Bk * threadIdx.x + index_k < N)
                {

                    tmpReduceMax =
                        max(tmpReduceMax, val[index_q * Bk + index_k]);
                }
            }
            __syncthreads();
            tmpReduceMax = WarpAllReduce<MaxOp, float, Bc>(tmpReduceMax);
            if (threadIdx.x == 0)
            {
                shareQK[threadIdx.y * Rq + index_q] = tmpReduceMax;
            }
            __syncthreads();
            float tmpReduceSum = 0.0f;
            for (int index_k = 0; index_k < Bk; index_k++)
            {
                if (indQ + threadIdx.y * Rq + index_q < N &&
                    indK + Bk * threadIdx.x + index_k < N)
                {
                    tmpReduceSum += __expf(val[index_q * Bk + index_k] -
                                           shareQK[threadIdx.y * Rq + index_q]);
                }
            }
            __syncthreads();
            tmpReduceSum = WarpAllReduce<SumOp, float, Bc>(tmpReduceSum);
            if (threadIdx.x == 0)
            {
                shareQK[threadIdx.y * Rq + index_q + numQ] = tmpReduceSum;
            }
            __syncthreads();
            if (newMax[index_q] > shareQK[threadIdx.y * Rq + index_q])
            {
                newSum[index_q] =
                    std::fma(shareQK[threadIdx.y * Rq + index_q + numQ],
                             __expf(shareQK[threadIdx.y * Rq + index_q] -
                                    newMax[index_q]),
                             newSum[index_q]);
            }
            else
            {
                newSum[index_q] =
                    std::fma(newSum[index_q],
                             __expf(newMax[index_q] -
                                    shareQK[threadIdx.y * Rq + index_q]),
                             shareQK[threadIdx.y * Rq + index_q + numQ]);

                newMax[index_q] = shareQK[threadIdx.y * Rq + index_q];
            }
            // PV
            for (int index_v = 0; index_v < Rv; index_v++)
            {
                sumSV[index_q * Rv + index_v] *=
                    __expf(oldMax[index_q] - newMax[index_q]);
            }
        }

        matmulSV(shareQK, inputV, shareVK, N, d, j, indQ, indK, indV, val,
                 newMax, sumSV);

        for (int index_q = 0; index_q < Rq; index_q++)
        {
            oldMax[index_q] = newMax[index_q];
        }

        __syncthreads();
    }
    for (int index_q = 0; index_q < Rq; index_q++)
    {
        float inv = __fdividef(1.0F, newSum[index_q]);
        for (int index_v = 0; index_v < Rv; index_v++)
        {
            sumSV[index_q * Rv + index_v] = sumSV[index_q * Rv + index_v] * inv;
        }
    }
    for (int index_q = 0; index_q < Rq; index_q++)
    {

        for (int id = 0; id < Rv; id += 4)
        {
            if (indQ + threadIdx.y * Rq + index_q < N &&
                indV + threadIdx.x * Rv + id < d)
            {
                (float4 &)output[(indQ + threadIdx.y * Rq + index_q) * d +
                                 indV + threadIdx.x * Rv + id] =
                    (float4 &)sumSV[index_q * Rv + id];
            }
        }
    }
}
void attention(float *cpu_Q, float *cpu_K, float *cpu_V, int N, int d, float *cpu_output)
{
    double st, ela;
    st = get_walltime();

    float *inputQ, *inputK, *inputV, *output;
    hipMalloc((void **)&inputQ, N * d * sizeof(float));
    hipMalloc((void **)&inputK, N * d * sizeof(float));
    hipMalloc((void **)&inputV, N * d * sizeof(float));

    hipMalloc((void **)&output, N * d * sizeof(float));
    hipMemcpy(inputQ, cpu_Q, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputK, cpu_K, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputV, cpu_V, N * d * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int num_block_x = (d + Rv * Bc - 1) / (Rv * Bc);
    int num_block_y = (N + Rq * Br - 1) / (Rq * Br);
    dim3 grid_dim(num_block_x, num_block_y, 1);
    dim3 block_dim(Bc, Br, 1);

    _attentionKernel<Br, Bc, Rq, Rv>
        <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        // Possibly: exit(-1) if program cannot continue....
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop); // must float ker_time
    hipMemcpy(cpu_output, output, N * d * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(inputQ);
    hipFree(inputK);
    hipFree(inputV);

    hipFree(output);

    ela = get_walltime() - st;

    printf("BlockReduce,kernel time:%.4f, use time:%.4f\n", ker_time / 1000., ela);
    // getThreadNum();
}
int main()
{
    int N = 1024;
    int d = 1024;

    int size = N * d;

    float *cpu_Q, *cpu_K, *cpu_V, *cpu_output;
    cpu_Q = (float *)malloc(size * sizeof(float));
    cpu_K = (float *)malloc(size * sizeof(float));
    cpu_V = (float *)malloc(size * sizeof(float));
    cpu_output = (float *)malloc(size * sizeof(float));
    for (int i = 0; i < size; i++)
    {
        cpu_Q[i] = i % 4;
        cpu_K[i] = i % 4;
        cpu_V[i] = i % 4;
        // printf("Q:%.4f\n",cpu_Q[i]);
    }

    attention(cpu_Q, cpu_K, cpu_V, N, d, cpu_output);
    for (int i = 0; i < 10; i++)
    {

        printf("out:%.6e\n", cpu_output[i]);
    }

    free(cpu_Q);
    free(cpu_K);
    free(cpu_V);
    free(cpu_output);

    return 0;
}



