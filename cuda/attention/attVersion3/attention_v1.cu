#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cub/block/block_reduce.cuh>
#include <>
void getThreadNum();
double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}

template <typename T>
struct SumOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return a + b;
    }
};

template <typename T>
struct MaxOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return max(a, b);
    }
};
template <template <typename> class ReductionOp, typename T,
          int thread_group_width = warpSize>
__inline__ __device__ T WarpAllReduce(T val)
{
    for (int mask = thread_group_width / 2; mask > 0; mask >>= 1)
    {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }

    return val;
}
template <int Br, int Bc, int Rq, int Rv>
__global__ void _attentionKernel(const float *__restrict inputQ,
                                 const float *__restrict inputK,
                                 const float *__restrict inputV, int N, int d,
                                 float *__restrict output)
{

    __shared__ float sumQK[Rq * Br * Bc];
    float sumSV[Rq * Rv];
    __shared__ float block_max[Rq][Br];
    __shared__ float block_sum[Rq][Br];

    int indV = Rv * (threadIdx.x + blockIdx.x * blockDim.x);
    int indQ = Rq * (threadIdx.y + blockIdx.y * blockDim.y);
    float newMax[Rq];
    float oldMax[Rq];
    float newSum[Rq];

    float out[Rq * Rv];
    int index_q = 0;
    int index_v = 0;
    for (index_q = 0; index_q < Rq; index_q++)
    {
        newMax[index_q] = -__FLT_MAX__;
        oldMax[index_q] = -__FLT_MAX__;
        newSum[index_q] = 0.0f;
        for (index_v = 0; index_v < Rv; index_v++)
        {
            out[index_q * Rv + index_v] = 0.0f;
        }
    }
    float regQK[Rq];
    float regTmp[Rq];
    int Tc = (N + Bc - 1) / Bc;

    for (int j = 0; j < Tc; j++)
    {

        int indK = threadIdx.x + j * Bc;

        for (index_q = 0; index_q < Rq; index_q++)
        {
            float val = 0.0f;
            for (int index = 0; index < d; index++)
            {
                val += inputQ[(indQ + index_q) * d + index] * inputK[indK * d + index];
            }
            if (indQ + index_q < N && indK < N)
            {

                regQK[index_q] = val;
                regTmp[index_q] = val;
            }
            else
            {

                regTmp[index_q] = -__FLT_MAX__;
            }
            __syncthreads();
        }

        // softmax reduce
        for (index_q = 0; index_q < Rq; index_q++)
        {
            regTmp[index_q] = WarpAllReduce<MaxOp, float, Bc>(regTmp[index_q]);
            if (threadIdx.x == 0)
            {
                block_max[index_q][threadIdx.y] = regTmp[index_q];
            }
            __syncthreads();
        }
        //--------------------
        for (index_q = 0; index_q < Rq; index_q++)
        {
            if (indQ + index_q < N && indK < N)
            {
                regTmp[index_q] = __expf(regQK[index_q] - block_max[index_q][threadIdx.y]);
            }
            else
            {

                regTmp[index_q] = 0.0f;
            }
            __syncthreads();
        }
        for (index_q = 0; index_q < Rq; index_q++)
        {
            regTmp[index_q] = WarpAllReduce<SumOp, float, Bc>(regTmp[index_q]);
            if (threadIdx.x == 0)
            {
                block_sum[index_q][threadIdx.y] = regTmp[index_q];
            }
            __syncthreads();
        }
        for (index_q = 0; index_q < Rq; index_q++)
        {
            if (newMax[index_q] > block_max[index_q][threadIdx.y])
            {
                newSum[index_q] = std::fma(block_sum[index_q][threadIdx.y], __expf(block_max[index_q][threadIdx.y] - newMax[index_q]), newSum[index_q]);
            }
            else
            {
                newSum[index_q] = std::fma(newSum[index_q], __expf(newMax[index_q] - block_max[index_q][threadIdx.y]), block_sum[index_q][threadIdx.y]);

                newMax[index_q] = block_max[index_q][threadIdx.y];
            }
        }

        for (index_q = 0; index_q < Rq; index_q++)
        {
            if (indQ + index_q < N && indK < N)
            {
                sumQK[(threadIdx.y * Rq + index_q) * Bc + threadIdx.x] = __expf(regQK[index_q] - newMax[index_q]);
            }
            else
            {

                sumQK[(threadIdx.y * Rq + index_q) * Bc + threadIdx.x] = 0.0f;
            }
            __syncthreads();
        }
        for (index_q = 0; index_q < Rq; index_q++)
        {
            for (index_v = 0; index_v < Rv; index_v++)
            {
                sumSV[index_q * Rv + index_v] = 0.0f;
                for (int phc = 0; phc < Bc; phc++)
                {
                    if (phc + j * Bc < N)
                    {
                        sumSV[index_q * Rv + index_v] += sumQK[(threadIdx.y * Rq + index_q) * Bc + phc] * inputV[(phc + j * Bc) * d + indV + index_v];
                    }
                }
            }
        }
        for (index_q = 0; index_q < Rq; index_q++)
        {
            for (index_v = 0; index_v < Rv; index_v++)
            {
                out[index_q * Rv + index_v] = std::fma(__expf(oldMax[index_q] - newMax[index_q]), out[index_q * Rv + index_v], sumSV[index_q * Rv + index_v]);
            }
        }

        for (index_q = 0; index_q < Rq; index_q++)
        {
            oldMax[index_q] = newMax[index_q];
        }

        __syncthreads();
    }
    for (index_q = 0; index_q < Rq; index_q++)
    {
        for (index_v = 0; index_v < Rv; index_v++)
        {
            if (indQ + index_q < N && indV + index_v < d)
            {
                output[(indQ + index_q) * d + indV + index_v] = out[index_q * Rv + index_v] * __fdividef(1.0F, newSum[index_q]);
            }
        }
    }
}
void attention(float *cpu_Q, float *cpu_K, float *cpu_V, int N, int d, float *cpu_output)
{
    double st, ela;
    st = get_walltime();

    float *inputQ, *inputK, *inputV, *output;
    hipMalloc((void **)&inputQ, N * d * sizeof(float));
    hipMalloc((void **)&inputK, N * d * sizeof(float));
    hipMalloc((void **)&inputV, N * d * sizeof(float));

    hipMalloc((void **)&output, N * d * sizeof(float));
    hipMemcpy(inputQ, cpu_Q, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputK, cpu_K, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputV, cpu_V, N * d * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int Br = 32;
    int Bc = 32; // Br>=Bc
    int Rq = 2;
    int Rv = 2;
    int num_block_x = (d + Rv * Bc - 1) / (Rv * Bc);
    int num_block_y = (N + Rq * Br - 1) / (Rq * Br);
    dim3 grid_dim(num_block_x, num_block_y, 1);
    dim3 block_dim(Bc, Br, 1);

    _attentionKernel<32, 32, 2, 2>
        <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        // Possibly: exit(-1) if program cannot continue....
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop); // must float ker_time
    hipMemcpy(cpu_output, output, N * d * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(inputQ);
    hipFree(inputK);
    hipFree(inputV);

    hipFree(output);

    ela = get_walltime() - st;

    printf("BlockReduce,kernel time:%.4f, use time:%.4f\n", ker_time / 1000., ela);
    // getThreadNum();
}
int main()
{
    int N = 4;
    int d = 3;

    int size = N * d;

    float *cpu_Q, *cpu_K, *cpu_V, *cpu_output;
    cpu_Q = (float *)malloc(size * sizeof(float));
    cpu_K = (float *)malloc(size * sizeof(float));
    cpu_V = (float *)malloc(size * sizeof(float));
    cpu_output = (float *)malloc(size * sizeof(float));
    for (int i = 0; i < size; i++)
    {
        cpu_Q[i] = i % 4;
        cpu_K[i] = i % 4;
        cpu_V[i] = i % 4;
        // printf("Q:%.4f\n",cpu_Q[i]);
    }

    attention(cpu_Q, cpu_K, cpu_V, N, d, cpu_output);
    for (int i = 0; i < 10; i++)
    {

        printf("out:%.6e\n", cpu_output[i]);
    }

    free(cpu_Q);
    free(cpu_K);
    free(cpu_V);
    free(cpu_output);

    return 0;
}
void getThreadNum()
{
    int deviceCount;

    hipGetDeviceCount(&deviceCount); // Returns in *deviceCount the number of devices
    printf("deviceCount: %d\n ", deviceCount);

    if (deviceCount == 0)
    {
        printf("error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }

    int dev = 0;
    hipSetDevice(dev); // Sets dev=0 device as the current device for the calling host thread.

    hipDeviceProp_t devProps;
    hipGetDeviceProperties(&devProps, dev);
    printf("name:%s\n", devProps.name);
    printf("totalGlobalMem: %ld\n", devProps.totalGlobalMem);
    printf("regsPerBlock: %d\n", devProps.regsPerBlock);
    printf("warpSize: %d\n", devProps.warpSize);
    printf("memPitch: %ld\n\n", devProps.memPitch);

    printf("一个线程块中可使用的最大共享内存\n");
    printf("devProps.sharedMemPerBlock: %ld Bytes \n\n", devProps.sharedMemPerBlock);

    printf("一个线程块中可包含的最大线程数量\n");
    printf("maxThreadsPerBlock: %d\n", devProps.maxThreadsPerBlock);

    printf("多维线程块数组中每一维可包含的最大线程数量\n");
    printf("maxThreadsDim[0]: %d\n", devProps.maxThreadsDim[0]);
    printf("maxThreadsDim[1]: %d\n", devProps.maxThreadsDim[1]);
    printf("maxThreadsDim[2]: %d\n\n", devProps.maxThreadsDim[2]);

    printf("一个线程格中每一维可包含的最大线程块数量\n");
    printf("maxGridSize[0]: %d\n", devProps.maxGridSize[0]);
    printf("maxGridSize[1]: %d\n", devProps.maxGridSize[1]);
    printf("maxGridSize[2]: %d\n\n", devProps.maxGridSize[2]);

    printf("clockRate: %d\n", devProps.clockRate);
    printf("totalConstMem: %ld\n", devProps.totalConstMem);
    printf("textureAlignment: %ld\n\n", devProps.textureAlignment);

    printf("计算能力：%d.%d\n", devProps.major, devProps.minor);
}

