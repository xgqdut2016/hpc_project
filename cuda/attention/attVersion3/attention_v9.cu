#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cub/block/block_reduce.cuh>
#include <>

double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}

const int Rq = 4;
const int Rv = 8; // 必须是4的倍数
const int Br = 16;
const int Bc = 16;
const int Bk = 4; // 必须是4的倍数
const int Bd = 4;
template <int Br, int Bc, int Rq>
__device__ void matmulRQK(const float *__restrict inputQ,
                          const float *__restrict inputK, float *shareQK,
                          float *shareVK, int N, int d, int width, int indQ,
                          int indK, float *val)
{
    float a[4];
    for (int ph = 0; ph < width; ph++)
    {
        for (int index_k = 0; index_k < Bk; index_k++)
        {
            (float4 &)a[0] = (float4 &)
                inputK[(indK + index_k) * d + (threadIdx.y + ph * Bc) * Bd];
            for (int id = 0; id < Bd; id++)
            {
                if (threadIdx.y < Bc)
                {
                    shareVK[(threadIdx.y * Bd + id) * Bc * Bk +
                            threadIdx.x * Bk + index_k] = a[id];
                    if (indK + index_k >= N ||
                        (threadIdx.y + ph * Bc) * Bd + id >= d)
                    {

                        shareVK[(threadIdx.y * Bd + id) * Bc * Bk +
                                threadIdx.x * Bk + index_k] = 0.0f;
                    }
                }
            }
        }

        for (int index_q = 0; index_q < Rq; index_q++)
        {
            (float4 &)shareQK[(threadIdx.y * Rq + index_q) * Bc * Bd +
                              threadIdx.x * Bd] = (float4 &)
                inputQ[(indQ + index_q) * d + (threadIdx.x + ph * Bc) * Bd];
            for (int id = 0; id < Bd; id++)
            {
                if (indQ + index_q >= N ||
                    (threadIdx.x + ph * Bc) * Bd + id >= d)
                {
                    shareQK[(threadIdx.y * Rq + index_q) * Bc * Bd +
                            threadIdx.x * Bd + id] = 0.0f;
                }
            }
        }
        __syncthreads();

        for (int index = 0; index < Bc * Bd; index++)
        {
            for (int index_q = 0; index_q < Rq; index_q++)
            {
                for (int index_k = 0; index_k < Bk; index_k++)
                {
                    val[index_q * Bk + index_k] = std::fma(
                        shareQK[(threadIdx.y * Rq + index_q) * Bc * Bd + index],
                        shareVK[index * Bc * Bk + threadIdx.x * Bk + index_k],
                        val[index_q * Bk + index_k]);
                }
            }
        }
        __syncthreads();
    }
}
template <int Br, int Bc, int Rq, int Rv>
__device__ void matmulSV(float *shareQK, const float *__restrict inputV,
                         float *shareVK, int N, int d, int j, int indQ,
                         int indK, int indV, float *val, float *newMax,
                         float *sumSV)
{
    if (threadIdx.y < Bc)
    {
        for (int index_k = 0; index_k < Bk; index_k++)
        {
            for (int id = 0; id < (int)(Rv / 4); id++)
            {
                (float4 &)shareVK[(threadIdx.y * Bk + index_k) * Bc * Rv +
                                  threadIdx.x * Rv + id * 4] = (float4 &)
                    inputV[((threadIdx.y + j * Bc) * Bk + index_k) * d + indV +
                           id * 4];
            }
            for (int index_v = 0; index_v < Rv; index_v++)
            {
                if ((threadIdx.y + j * Bc) * Bk + index_k >= N ||
                    indV + index_v >= d)
                {
                    shareVK[(threadIdx.y * Bk + index_k) * Bc * Rv +
                            threadIdx.x * Rv + index_v] = 0.0f;
                }
            }
        }
    }
    for (int index_q = 0; index_q < Rq; index_q++)
    {
        for (int index_k = 0; index_k < Bk; index_k++)
        {
            if (indQ + index_q < N && indK + index_k < N)
            {
                shareQK[(threadIdx.y * Rq + index_q) * Bc * Bk +
                        threadIdx.x * Bk + index_k] =
                    __expf(val[index_q * Bk + index_k] - newMax[index_q]);
            }
            else
            {

                shareQK[(threadIdx.y * Rq + index_q) * Bc * Bk +
                        threadIdx.x * Bk + index_k] = 0.0f;
            }
        }
    }
    __syncthreads();

    for (int phc = 0; phc < Bc * Bk; phc++)
    {
        for (int index_q = 0; index_q < Rq; index_q++)
        {

            for (int index_v = 0; index_v < Rv; index_v++)
            {
                sumSV[index_q * Rv + index_v] +=
                    shareQK[(threadIdx.y * Rq + index_q) * Bc * Bk + phc] *
                    shareVK[phc * Bc * Rv + threadIdx.x * Rv + index_v];
            }
        }
    }
}
template <typename T>
struct SumOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return a + b;
    }
};

template <typename T>
struct MaxOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return max(a, b);
    }
};
template <template <typename> class ReductionOp, typename T,
          int thread_group_width = 32>
__inline__ __device__ T WarpAllReduce(T val)
{
    for (int mask = thread_group_width / 2; mask > 0; mask >>= 1)
    {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }

    return val;
}

template <int Br, int Bc, int Rq, int Rv>
__global__ void _attentionKernel(const float *__restrict inputQ,
                                 const float *__restrict inputK,
                                 const float *__restrict inputV, int N, int d,
                                 float *__restrict output)
{

    __shared__ float shareQK[Rq * Br * Bc * Bk];
    __shared__ float shareVK[Bk * Bc * Bc * Rv];

    float sumSV[Rq * Rv] = {0.0f};
    float newMax[Rq];
    float oldMax[Rq];
    float newSum[Rq] = {0.0f};

    float val[Rq * Bk];

    int indV = Rv * (threadIdx.x + blockIdx.x * blockDim.x);
    int indQ = Rq * (threadIdx.y + blockIdx.y * blockDim.y);

    for (int index_q = 0; index_q < Rq; index_q++)
    {
        newMax[index_q] = -__FLT_MAX__;
        oldMax[index_q] = -__FLT_MAX__;
    }

    int Tc = (N + Bc * Bk - 1) / (Bc * Bk);

    int width = (d + Bc * Bd - 1) / (Bc * Bd);
    for (int j = 0; j < Tc; j++)
    {

        int indK = Bk * (threadIdx.x + j * Bc);
        for (int index_q = 0; index_q < Rq; index_q++)
        {
            for (int index_k = 0; index_k < Bk; index_k++)
            {

                val[index_q * Bk + index_k] = 0.0f;
            }
        }
        matmulRQK<Br, Bc, Rq>(inputQ, inputK, shareQK, shareVK, N, d, width,
                              indQ, indK, val);
        for (int index_q = 0; index_q < Rq; index_q++)
        {
            float tmpReduceMax = -__FLT_MAX__;
            for (int index_k = 0; index_k < Bk; index_k++)
            {
                if (indQ + index_q < N && indK + index_k < N)
                {

                    tmpReduceMax =
                        max(tmpReduceMax, val[index_q * Bk + index_k]);
                }
            }
            __syncthreads();
            tmpReduceMax = WarpAllReduce<MaxOp, float, Bc>(tmpReduceMax);
            if (threadIdx.x == 0)
            {
                shareQK[threadIdx.y * Rq + index_q] = tmpReduceMax;
            }
            __syncthreads();
            float tmpReduceSum = 0.0f;
            for (int index_k = 0; index_k < Bk; index_k++)
            {
                if (indQ + index_q < N && indK + index_k < N)
                {
                    tmpReduceSum += __expf(val[index_q * Bk + index_k] -
                                           shareQK[threadIdx.y * Rq + index_q]);
                }
            }
            __syncthreads();
            tmpReduceSum = WarpAllReduce<SumOp, float, Bc>(tmpReduceSum);
            if (threadIdx.x == 0)
            {
                shareQK[threadIdx.y * Rq + index_q + Rq * Br] = tmpReduceSum;
            }
            __syncthreads();
            if (newMax[index_q] > shareQK[threadIdx.y * Rq + index_q])
            {
                newSum[index_q] =
                    std::fma(shareQK[threadIdx.y * Rq + index_q + Rq * Br],
                             __expf(shareQK[threadIdx.y * Rq + index_q] -
                                    newMax[index_q]),
                             newSum[index_q]);
            }
            else
            {
                newSum[index_q] =
                    std::fma(newSum[index_q],
                             __expf(newMax[index_q] -
                                    shareQK[threadIdx.y * Rq + index_q]),
                             shareQK[threadIdx.y * Rq + index_q + Rq * Br]);

                newMax[index_q] = shareQK[threadIdx.y * Rq + index_q];
            }
            // PV
            for (int index_v = 0; index_v < Rv; index_v++)
            {
                sumSV[index_q * Rv + index_v] *=
                    __expf(oldMax[index_q] - newMax[index_q]);
            }
        }

        matmulSV<Br, Bc, Rq, Rv>(shareQK, inputV, shareVK, N, d, j, indQ, indK,
                                 indV, val, newMax, sumSV);

        for (int index_q = 0; index_q < Rq; index_q++)
        {
            oldMax[index_q] = newMax[index_q];
        }

        //__syncthreads();
    }
    for (int index_q = 0; index_q < Rq; index_q++)
    {
        float inv = __fdividef(1.0F, newSum[index_q]);
        for (int index_v = 0; index_v < Rv; index_v++)
        {
            sumSV[index_q * Rv + index_v] = sumSV[index_q * Rv + index_v] * inv;
        }
    }
    for (int index_q = 0; index_q < Rq; index_q++)
    {

        for (int id = 0; id < (int)(Rv / 4); id++)
        {
            if (indQ + index_q < N)
            {
                (float4 &)output[(indQ + index_q) * d + indV + id * 4] =
                    (float4 &)sumSV[index_q * Rv + id * 4];
            }
        }
    }
}
void attention(float *cpu_Q, float *cpu_K, float *cpu_V, int N, int d, float *cpu_output)
{
    double st, ela;
    st = get_walltime();

    float *inputQ, *inputK, *inputV, *output;
    hipMalloc((void **)&inputQ, N * d * sizeof(float));
    hipMalloc((void **)&inputK, N * d * sizeof(float));
    hipMalloc((void **)&inputV, N * d * sizeof(float));

    hipMalloc((void **)&output, N * d * sizeof(float));
    hipMemcpy(inputQ, cpu_Q, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputK, cpu_K, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputV, cpu_V, N * d * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int num_block_x = (d + Rv * Bc - 1) / (Rv * Bc);
    int num_block_y = (N + Rq * Br - 1) / (Rq * Br);
    dim3 grid_dim(num_block_x, num_block_y, 1);
    dim3 block_dim(Bc, Br, 1);

    _attentionKernel<Br, Bc, Rq, Rv>
        <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        // Possibly: exit(-1) if program cannot continue....
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop); // must float ker_time
    hipMemcpy(cpu_output, output, N * d * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(inputQ);
    hipFree(inputK);
    hipFree(inputV);

    hipFree(output);

    ela = get_walltime() - st;

    printf("BlockReduce,kernel time:%.4f, use time:%.4f\n", ker_time / 1000., ela);
}
int main()
{
    int N = 1024;
    int d = 1024;

    int size = N * d;

    float *cpu_Q, *cpu_K, *cpu_V, *cpu_output;
    cpu_Q = (float *)malloc(size * sizeof(float));
    cpu_K = (float *)malloc(size * sizeof(float));
    cpu_V = (float *)malloc(size * sizeof(float));
    cpu_output = (float *)malloc(size * sizeof(float));
    for (int i = 0; i < size; i++)
    {
        cpu_Q[i] = i % 4;
        cpu_K[i] = i % 4;
        cpu_V[i] = i % 4;
        // printf("Q:%.4f\n",cpu_Q[i]);
    }

    attention(cpu_Q, cpu_K, cpu_V, N, d, cpu_output);
    for (int i = 0; i < 10; i++)
    {

        printf("out:%.6e\n", cpu_output[i]);
    }

    free(cpu_Q);
    free(cpu_K);
    free(cpu_V);
    free(cpu_output);

    return 0;
}


