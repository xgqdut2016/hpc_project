#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cub/block/block_reduce.cuh>
#include <>
void getThreadNum();

double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}

const int Rq = 2;
const int Rv = 1;
const int Rate = 4;
const int Br = 32;
const int Bc = 32;

template <int Br, int Bc, int Rq>
__device__ void matmulRQK(const float *__restrict inputQ,
                          const float *__restrict inputK, float *shareQK,
                          float *shareVK, int N, int d, int width, int indQ,
                          int indK, float *regLeft, float *val)
{
    float a[4];
    for (int ph = 0; ph < width; ph++)
    {
        if (threadIdx.y < Bc)
        {
            (float4 &)a[0] =
                (float4 &)inputK[indK * d + (threadIdx.y + ph * Bc) * Rate];
            for (int id = 0; id < Rate; id++)
            {
                shareVK[(threadIdx.y * Rate + id) * Bc + threadIdx.x] = a[id];
                if (indK >= N || (threadIdx.y + ph * Bc) * Rate + id >= d)
                {
                    shareVK[(threadIdx.y * Rate + id) * Bc + threadIdx.x] =
                        0.0f;
                }
            }
            // for (int id = 0; id < Rate; id++) {
            //     shareVK[(threadIdx.y * Rate + id) * Bc + threadIdx.x] = 0.0f;
            //     if (indK < N || (threadIdx.y + ph * Bc) * Rate + id < d) {
            //         shareVK[(threadIdx.y * Rate + id) * Bc + threadIdx.x] =
            //             inputK[indK * d + (threadIdx.y + ph * Bc) * Rate +
            //             id];
            //     }
            // }
        }

        for (int index_q = 0; index_q < Rq; index_q++)
        {
            (float4 &)shareQK[(threadIdx.y * Rq + index_q) * Bc * Rate +
                              threadIdx.x * Rate] = (float4 &)
                inputQ[(indQ + index_q) * d + (threadIdx.x + ph * Bc) * Rate];
            for (int id = 0; id < Rate; id++)
            {
                if (indQ + index_q >= N ||
                    (threadIdx.x + ph * Bc) * Rate + id >= d)
                {
                    shareQK[(threadIdx.y * Rq + index_q) * Bc * Rate +
                            threadIdx.x * Rate] = 0.0f;
                }
            }
            // for (int id = 0; id < Rate; id++) {
            //     if (indQ + index_q < N &&
            //         (threadIdx.x + ph * Bc) * Rate + id < d) {
            //         shareQK[(threadIdx.y * Rq + index_q) * Bc * Rate +
            //                 threadIdx.x * Rate + id] =
            //             inputQ[(indQ + index_q) * d +
            //                    (threadIdx.x + ph * Bc) * Rate + id];
            //     } else {
            //         shareQK[(threadIdx.y * Rq + index_q) * Bc * Rate +
            //                 threadIdx.x * Rate + id] = 0.0f;
            //     }
            // }
        }
        __syncthreads();
        for (int index = 0; index < Bc * Rate; index++)
        {
            for (int index_q = 0; index_q < Rq; index_q++)
            {
                regLeft[index_q] =
                    shareQK[(threadIdx.y * Rq + index_q) * Bc * Rate + index];
                val[index_q] =
                    fmaf(regLeft[index_q], shareVK[index * Bc + threadIdx.x],
                         val[index_q]);
            }
        }
        __syncthreads();
    }
}
template <int Br, int Bc, int Rq, int Rv>
__device__ void matmulSV(float *shareQK, const float *__restrict inputV,
                         float *shareVK, int N, int d, int j, int indQ,
                         int indK, int indV, float *val, float *newMax,
                         float *regLeft, float *regRight, float *sumSV)
{

    if (threadIdx.y < Bc)
    {
        for (int index_v = 0; index_v < Rv; index_v++)
        {
            (float4 &)shareVK[threadIdx.y * Bc * Rv * Rate +
                              (threadIdx.x * Rv + index_v) * Rate] = (float4 &)
                inputV[(threadIdx.y + j * Bc) * d + indV + index_v * Rate];
            for (int id = 0; id < Rate; id++)
            {
                if (threadIdx.y + j * Bc >= N ||
                    indV + index_v * Rate + id >= d)
                {
                    shareVK[threadIdx.y * Bc * Rv * Rate +
                            (threadIdx.x * Rv + index_v) * Rate + id] = 0.0f;
                }
            }
        }
    }
    for (int index_q = 0; index_q < Rq; index_q++)
    {
        if (indQ + index_q < N && indK < N)
        {
            shareQK[(threadIdx.y * Rq + index_q) * Bc + threadIdx.x] =
                __expf(val[index_q] - newMax[index_q]);
        }
        else
        {

            shareQK[(threadIdx.y * Rq + index_q) * Bc + threadIdx.x] = 0.0f;
        }
    }
    __syncthreads();

    for (int phc = 0; phc < Bc; phc++)
    {

        for (int index_q = 0; index_q < Rq; index_q++)
        {

            for (int index_v = 0; index_v < Rv; index_v++)
            {
                for (int id = 0; id < Rate; id++)
                {
                    sumSV[index_q * Rv * Rate + index_v * Rate + id] =
                        fmaf(shareQK[(threadIdx.y * Rq + index_q) * Bc + phc],
                             shareVK[phc * Bc * Rv * Rate +
                                     (threadIdx.x * Rv + index_v) * Rate + id],
                             sumSV[index_q * Rv * Rate + index_v * Rate + id]);
                }
            }
        }
    }
}
template <typename T>
struct SumOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return a + b;
    }
};

template <typename T>
struct MaxOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return max(a, b);
    }
};
template <template <typename> class ReductionOp, typename T,
          int thread_group_width = warpSize>
__inline__ __device__ T WarpAllReduce(T val)
{
    for (int mask = thread_group_width / 2; mask > 0; mask >>= 1)
    {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }

    return val;
}

template <int Br, int Bc, int Rq, int Rv>
__global__ void _attentionKernel(const float *__restrict inputQ,
                                 const float *__restrict inputK,
                                 const float *__restrict inputV, int N, int d,
                                 float *__restrict output)
{

    __shared__ float shareQK[Rq * Br * Bc * Rate];
    __shared__ float shareVK[Bc * Bc * Rv * Rate];

    float sumSV[Rq * Rv * Rate] = {0.0f};

    float newMax[Rq];
    float oldMax[Rq];
    float newSum[Rq];
    float regLeft[Rq];
    float regRight[Rv];
    float val[Rq];
    float regTmp[Rq];

    int indV = Rate * Rv * (threadIdx.x + blockIdx.x * blockDim.x);
    int indQ = Rq * (threadIdx.y + blockIdx.y * blockDim.y);

    for (int index_q = 0; index_q < Rq; index_q++)
    {
        newMax[index_q] = -__FLT_MAX__;
        oldMax[index_q] = -__FLT_MAX__;
        newSum[index_q] = 0.0f;
    }

    int Tc = (N + Bc - 1) / Bc;

    int width = (d + Bc * Rate - 1) / (Bc * Rate);
    for (int j = 0; j < Tc; j++)
    {

        int indK = threadIdx.x + j * Bc;
        for (int index_q = 0; index_q < Rq; index_q++)
        {
            val[index_q] = 0.0f;
        }

        matmulRQK<Br, Bc, Rq>(inputQ, inputK, shareQK, shareVK, N, d, width,
                              indQ, indK, regLeft, val);
        for (int index_q = 0; index_q < Rq; index_q++)
        {
            if (indQ + index_q < N && indK < N)
            {

                regTmp[index_q] = val[index_q];
            }
            else
            {

                regTmp[index_q] = -__FLT_MAX__;
            }
        }
        __syncthreads();
        // softmax reduce
        for (int index_q = 0; index_q < Rq; index_q++)
        {
            regTmp[index_q] = WarpAllReduce<MaxOp, float, Bc>(regTmp[index_q]);
            if (threadIdx.x == 0)
            {
                shareQK[threadIdx.y * Rq + index_q] = regTmp[index_q];
            }
        }
        __syncthreads();
        //--------------------
        for (int index_q = 0; index_q < Rq; index_q++)
        {
            if (indQ + index_q < N && indK < N)
            {
                regTmp[index_q] =
                    __expf(val[index_q] - shareQK[threadIdx.y * Rq + index_q]);
            }
            else
            {

                regTmp[index_q] = 0.0f;
            }
        }
        __syncthreads();
        for (int index_q = 0; index_q < Rq; index_q++)
        {
            regTmp[index_q] = WarpAllReduce<SumOp, float, Bc>(regTmp[index_q]);
            if (threadIdx.x == 0)
            {
                shareQK[threadIdx.y * Rq + index_q + Rq * Br] = regTmp[index_q];
            }
        }
        __syncthreads();
        for (int index_q = 0; index_q < Rq; index_q++)
        {
            if (newMax[index_q] > shareQK[threadIdx.y * Rq + index_q])
            {
                newSum[index_q] =
                    fmaf(shareQK[threadIdx.y * Rq + index_q + Rq * Br],
                         __expf(shareQK[threadIdx.y * Rq + index_q] -
                                newMax[index_q]),
                         newSum[index_q]);
            }
            else
            {
                newSum[index_q] =
                    fmaf(newSum[index_q],
                         __expf(newMax[index_q] -
                                shareQK[threadIdx.y * Rq + index_q]),
                         shareQK[threadIdx.y * Rq + index_q + Rq * Br]);

                newMax[index_q] = shareQK[threadIdx.y * Rq + index_q];
            }
        }
        for (int index_q = 0; index_q < Rq; index_q++)
        {
            for (int index_v = 0; index_v < Rv; index_v++)
            {
                for (int id = 0; id < Rate; id++)
                {
                    sumSV[index_q * Rv * Rate + index_v * Rate + id] *=
                        __expf(oldMax[index_q] - newMax[index_q]);
                }
            }
        }
        matmulSV<Br, Bc, Rq, Rv>(shareQK, inputV, shareVK, N, d, j, indQ, indK,
                                 indV, val, newMax, regLeft, regRight, sumSV);

        for (int index_q = 0; index_q < Rq; index_q++)
        {
            oldMax[index_q] = newMax[index_q];
        }

        //__syncthreads();
    }
    for (int index_q = 0; index_q < Rq; index_q++)
    {
        float inv = __fdividef(1.0F, newSum[index_q]);
        for (int index_v = 0; index_v < Rv; index_v++)
        {
            for (int id = 0; id < Rate; id++)
            {
                sumSV[index_q * Rv * Rate + index_v * Rate + id] =
                    sumSV[index_q * Rv * Rate + index_v * Rate + id] * inv;
            }
        }
    }
    for (int index_q = 0; index_q < Rq; index_q++)
    {
        for (int index_v = 0; index_v < Rv; index_v++)
        {
            (float4 &)output[(indQ + index_q) * d + indV + index_v * Rate] =
                (float4 &)sumSV[index_q * Rv * Rate + index_v * Rate];
            // for (int id = 0; id < Rate; id++) {
            //     if (indQ + index_q < N && indV + index_v * Rate + id < d) {
            //         output[(indQ + index_q) * d + indV + index_v * Rate + id]
            //         =
            //             sumSV[index_q * Rv * Rate + index_v * Rate + id] *
            //             inv;
            //     }
            // }
        }
    }
}
void attention(float *cpu_Q, float *cpu_K, float *cpu_V, int N, int d, float *cpu_output)
{
    double st, ela;
    st = get_walltime();

    float *inputQ, *inputK, *inputV, *output;
    hipMalloc((void **)&inputQ, N * d * sizeof(float));
    hipMalloc((void **)&inputK, N * d * sizeof(float));
    hipMalloc((void **)&inputV, N * d * sizeof(float));

    hipMalloc((void **)&output, N * d * sizeof(float));
    hipMemcpy(inputQ, cpu_Q, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputK, cpu_K, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inputV, cpu_V, N * d * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int num_block_x = (d + Rv * Bc * Rate - 1) / (Rv * Bc * Rate);
    int num_block_y = (N + Rq * Br - 1) / (Rq * Br);
    dim3 grid_dim(num_block_x, num_block_y, 1);
    dim3 block_dim(Bc, Br, 1);

    _attentionKernel<Br, Bc, Rq, Rv>
        <<<grid_dim, block_dim>>>(inputQ, inputK, inputV, N, d, output);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        // Possibly: exit(-1) if program cannot continue....
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop); // must float ker_time
    hipMemcpy(cpu_output, output, N * d * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(inputQ);
    hipFree(inputK);
    hipFree(inputV);

    hipFree(output);

    ela = get_walltime() - st;

    printf("BlockReduce,kernel time:%.4f, use time:%.4f\n", ker_time / 1000., ela);
    // getThreadNum();
}
int main()
{
    int N = 3;
    int d = 128;

    int size = N * d;

    float *cpu_Q, *cpu_K, *cpu_V, *cpu_output;
    cpu_Q = (float *)malloc(size * sizeof(float));
    cpu_K = (float *)malloc(size * sizeof(float));
    cpu_V = (float *)malloc(size * sizeof(float));
    cpu_output = (float *)malloc(size * sizeof(float));
    for (int i = 0; i < size; i++)
    {
        cpu_Q[i] = i % 4;
        cpu_K[i] = i % 4;
        cpu_V[i] = i % 4;
        // printf("Q:%.4f\n",cpu_Q[i]);
    }

    attention(cpu_Q, cpu_K, cpu_V, N, d, cpu_output);
    for (int i = 0; i < 10; i++)
    {

        printf("out:%.6e\n", cpu_output[i]);
    }

    free(cpu_Q);
    free(cpu_K);
    free(cpu_V);
    free(cpu_output);

    return 0;
}
void getThreadNum()
{
    int deviceCount;

    hipGetDeviceCount(&deviceCount); // Returns in *deviceCount the number of devices
    printf("deviceCount: %d\n ", deviceCount);

    if (deviceCount == 0)
    {
        printf("error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }

    int dev = 0;
    hipSetDevice(dev); // Sets dev=0 device as the current device for the calling host thread.

    hipDeviceProp_t devProps;
    hipGetDeviceProperties(&devProps, dev);
    printf("name:%s\n", devProps.name);
    printf("totalGlobalMem: %ld\n", devProps.totalGlobalMem);
    printf("regsPerBlock: %d\n", devProps.regsPerBlock);
    printf("warpSize: %d\n", devProps.warpSize);
    printf("memPitch: %ld\n\n", devProps.memPitch);

    printf("一个线程块中可使用的最大共享内存\n");
    printf("devProps.sharedMemPerBlock: %ld Bytes \n\n", devProps.sharedMemPerBlock);

    printf("一个线程块中可包含的最大线程数量\n");
    printf("maxThreadsPerBlock: %d\n", devProps.maxThreadsPerBlock);

    printf("多维线程块数组中每一维可包含的最大线程数量\n");
    printf("maxThreadsDim[0]: %d\n", devProps.maxThreadsDim[0]);
    printf("maxThreadsDim[1]: %d\n", devProps.maxThreadsDim[1]);
    printf("maxThreadsDim[2]: %d\n\n", devProps.maxThreadsDim[2]);

    printf("一个线程格中每一维可包含的最大线程块数量\n");
    printf("maxGridSize[0]: %d\n", devProps.maxGridSize[0]);
    printf("maxGridSize[1]: %d\n", devProps.maxGridSize[1]);
    printf("maxGridSize[2]: %d\n\n", devProps.maxGridSize[2]);

    printf("clockRate: %d\n", devProps.clockRate);
    printf("totalConstMem: %ld\n", devProps.totalConstMem);
    printf("textureAlignment: %ld\n\n", devProps.textureAlignment);

    printf("计算能力：%d.%d\n", devProps.major, devProps.minor);
}


