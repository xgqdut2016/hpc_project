#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}
void initCpu(float *hostA, float *hostB, int n)
{
    for (int i = 0; i < n; i++)
    {
        hostA[i] = 1;
        hostB[i] = 1;
    }
}
void addCpu(float *hostA, float *hostB, float *hostC, int n)
{
    for (int i = 0; i < n; i++)
    {
        hostC[i] = hostA[i] + hostB[i];
    }
}
__global__ void addKernel(float *deviceA, float *deviceB, float *deviceC, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x; // 计算全局索引
    if (index < n)
    {
        deviceC[index] = deviceA[index] + deviceB[index];
    }
}
int main()
{
    float *hostA, *hostB, *hostC, *serialC;
    int n = 102400;

    hostA = (float *)malloc(n * sizeof(float));
    hostB = (float *)malloc(n * sizeof(float));
    hostC = (float *)malloc(n * sizeof(float));
    serialC = (float *)malloc(n * sizeof(float));
    initCpu(hostA, hostB, n);
    double stC, elaC;
    stC = get_walltime();
    addCpu(hostA, hostB, serialC, n);
    elaC = get_walltime() - stC;
    double st, ela;
    st = get_walltime();

    float *dA, *dB, *dC;
    hipMalloc((void **)&dA, n * sizeof(float));
    hipMalloc((void **)&dB, n * sizeof(float));
    hipMalloc((void **)&dC, n * sizeof(float));

    hipMemcpy(dA, hostA, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hostB, n * sizeof(float), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    int BLOCK_DIM = 1024;
    int num_block_x = n / BLOCK_DIM;
    int num_block_y = 1;
    dim3 grid_dim(num_block_x, num_block_y, 1);
    dim3 block_dim(BLOCK_DIM, 1, 1);
    addKernel<<<grid_dim, block_dim>>>(dA, dB, dC, n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&ker_time, start, stop); // must float ker_time
    hipMemcpy(hostC, dC, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    ela = get_walltime() - st;
    printf("n = %d: \n CPU use time:%.4f\n GPU use time:%.4f\n kernel time:%.4f\n", n, elaC, ela, ker_time / 1000.0);
    free(hostA);
    free(hostB);
    free(hostC);
    free(serialC);
    return 0;
}