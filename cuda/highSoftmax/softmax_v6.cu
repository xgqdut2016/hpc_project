#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cub/block/block_reduce.cuh>
#include <>
double get_walltime()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double)(tp.tv_sec + tp.tv_usec * 1e-6);
}
#define warpSize 32
#define max_function(a, b) ((a) > (b) ? (a) : (b))

struct __align__(8) MD
{                  // update the global max and sum, store the output at
                   // max_tmp and sum_tmp
    float max_tmp; // store max
    float sum_tmp; // store sum
};
__device__ __forceinline__ MD reduce_md_op(MD a, MD b)
{
    bool a_bigger = (a.max_tmp > b.max_tmp);
    MD bigger = a_bigger ? a : b;
    MD smaller = a_bigger ? b : a;
    MD res;
    res.sum_tmp = bigger.sum_tmp +
                  smaller.sum_tmp * __expf(smaller.max_tmp - bigger.max_tmp);
    res.max_tmp = bigger.max_tmp;
    return res;
}
template <int BLOCK_DIM>
__launch_bounds__(BLOCK_DIM) __global__ void _blockSoftmaxKernel(
    float *__restrict input, float *__restrict output, int size, int dimsize,
    int stride)
{ // if set axis = 1, inputShape=[I,J,K,S]
  // tid = i(JKS) + j(KS) + k(S) + s

    // blockDim.x = size/dimsize = IKS
    // blockIdx.x = i(KS) + k(S) + s,blockIdx.x%stride = k(S) + s

    int tid =
        blockIdx.x % stride + (blockIdx.x - blockIdx.x % stride) *
                                  dimsize; // now, tid = i(JKS) + k(S) + s;

    MD md_partial;
    md_partial.max_tmp = -__FLT_MAX__;
    md_partial.sum_tmp = 0.0f;
    MD md_input;
    for (int ph = 0; threadIdx.x + ph * BLOCK_DIM < dimsize; ph++)
    {

        md_input.max_tmp = input[tid + (threadIdx.x + ph * BLOCK_DIM) * stride];

        md_input.sum_tmp = 1.0f;
        md_partial = reduce_md_op(md_partial,
                                  md_input); // reduce the data to one block
    }
    typedef hipcub::BlockReduce<MD, BLOCK_DIM> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ MD md_total;
    MD md_block = BlockReduce(temp_storage).Reduce(md_partial, reduce_md_op);
    if (threadIdx.x ==
        0)
    { // must set threadIdx.x = 0 write the output to memory
        md_total = md_block;
    }
    __syncthreads();
    //-----------------

    for (int ph = 0; threadIdx.x + ph * BLOCK_DIM < dimsize; ph++)
    {
        output[tid + (threadIdx.x + ph * BLOCK_DIM) * stride] =
            __expf(input[tid + (threadIdx.x + ph * BLOCK_DIM) * stride] -
                   md_total.max_tmp) *
            __fdividef(1.0F, md_total.sum_tmp);
    }
}

template <typename T>
struct SumOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return a + b;
    }
};

template <typename T>
struct MaxOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return max(a, b);
    }
};
template <template <typename> class ReductionOp, typename T,
          int thread_group_width = warpSize>
__inline__ __device__ T WarpAllReduce(T val)
{
    for (int mask = thread_group_width / 2; mask > 0; mask /= 2)
    {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}

__global__ void _warpSoftmaxKernel(float *__restrict input,
                                   float *__restrict output, int size,
                                   int dimsize, int stride)
{
    int otherIdx = blockIdx.x * blockDim.y + threadIdx.y;
    int otherSize = size / dimsize;
    int tid = otherIdx % stride + (otherIdx - otherIdx % stride) * dimsize;
    // int num_per_thread = (dimsize + warpSize - 1) / warpSize;
    float buff[32];
    if (otherIdx < otherSize)
    {
        for (int ph = 0; threadIdx.x + ph * warpSize < dimsize; ph++)
        {
            buff[ph] = input[tid + (threadIdx.x + ph * warpSize) * stride];
        }
    }
    __shared__ float max_total[warpSize];
    __shared__ float sum_total[warpSize];
    float max_data = -__FLT_MAX__;
    if (otherIdx < otherSize)
    {
        for (int ph = 0; threadIdx.x + ph * warpSize < dimsize; ph++)
        {
            max_data = max_function(max_data, buff[ph]);
        }
    }

    max_data = WarpAllReduce<MaxOp, float, warpSize>(max_data);

    if (threadIdx.x == 0)
        max_total[threadIdx.y] = max_data;
    //__syncthreads();
    //--------------------------------------------
    float sum_data = 0.0f;
    if (otherIdx < otherSize)
    {
        for (int ph = 0; threadIdx.x + ph * warpSize < dimsize; ph++)
        {
            sum_data += __expf(buff[ph] - max_total[threadIdx.y]);
        }
    }
    sum_data = WarpAllReduce<SumOp, float, warpSize>(sum_data);

    if (threadIdx.x == 0)
        sum_total[threadIdx.y] = sum_data;
    //__syncthreads();
    //--------------------------------------------
    if (otherIdx < otherSize)
    {
        for (int ph = 0; threadIdx.x + ph * warpSize < dimsize; ph++)
        {
            output[tid + (threadIdx.x + ph * warpSize) * stride] =
                __expf(buff[ph] - max_total[threadIdx.y]) *
                __fdividef(1.0F, sum_total[threadIdx.y]);
        }
    }
}

void softmax(float *cpu_input, int size, int *cpu_inputShape, int axis, int nDims)
{
    double st, ela;
    st = get_walltime();
    int dimsize = cpu_inputShape[axis];
    int num_blocks = size / dimsize;
    float *input;
    hipMalloc((void **)&input, size * sizeof(float));

    hipMemcpy(input, cpu_input, size * sizeof(float), hipMemcpyHostToDevice);
    int *inputShape;
    hipMalloc((void **)&inputShape, nDims * sizeof(float));
    hipMemcpy(inputShape, cpu_inputShape, nDims * sizeof(float), hipMemcpyHostToDevice);
    int stride = 1, temp = 1; // stride=[JKS, KS, S, 1][axis]

    for (int i = nDims - 1; i >= 0; --i)
    { // must i = nDims - 1, --i; can't i = 0, i++
        if (i == axis)
        {
            stride = temp;
        }
        temp *= cpu_inputShape[i];
    }

    hipEvent_t start, stop;
    float ker_time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    if (dimsize > 1024)
    {
        int BLOCK_DIM = 1024;
        _blockSoftmaxKernel<1024>
            <<<num_blocks, BLOCK_DIM>>>(input, input, size, dimsize, stride);
    }
    else
    {

        int num_block_x = (num_blocks + warpSize - 1) / warpSize;
        dim3 block_dim(warpSize, warpSize, 1);
        dim3 grid_dim(num_block_x, 1, 1);

        _warpSoftmaxKernel<<<grid_dim, block_dim>>>(input, input, size,
                                                    dimsize, stride);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop); // must float ker_time
    hipMemcpy(cpu_input, input, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(input);
    hipFree(inputShape);

    ela = get_walltime() - st;

    printf("BlockReduce,kernel time:%.4f, use time:%.4f\n", ker_time / 1000., ela);
}
int main()
{

    int nDims = 2;
    int cpu_inputShape[nDims] = {2, 5};
    int axis = 1;

    int size = 1;
    for (int i = nDims - 1; i >= 0; --i)
    {
        size *= cpu_inputShape[i];
    }
    // printf("stride:%d\n",stride);

    float cpu_input[size] = {0, 1, 2, 3, 4, 10000, 10001, 10002, 10003, 10004};
    softmax(cpu_input, size, cpu_inputShape, axis, nDims);

    float s = 0;
    for (int i = 0; i < size; i++)
    {
        s += cpu_input[i];
        printf("softmax:%.4e\n", cpu_input[i]);
    }
    printf("s:%.3e\n", s);

    return 0;
}


